#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <stdexcept>

#include "allocator/allocatorFactory.h"
#include "device.h"
#include "helpers.h"
#include "engine/ops.h"
#include "tensor.h"

#define CUDA_CHECK(err)                                                \
  {                                                                    \
    hipError_t err_ = (err);                                          \
    if (err_ != hipSuccess) {                                         \
      throw std::runtime_error("CUDA Error: " +                        \
                               std::string(hipGetErrorString(err_))); \
    }                                                                  \
  }


Tensor CudaOps::pow(const Tensor& base, float exponent) {
  // TODO: pow gpu implemenation
}


Tensor CudaOps::pow(const Tensor& base, const Tensor& exponent) {
  // TODO: pow gpu implemenation
}

