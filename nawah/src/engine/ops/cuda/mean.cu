#include <hip/hip_runtime.h>

#include <stdexcept>

#include "allocator/allocatorFactory.h"
#include "device.h"
#include "helpers.h"
#include "engine/ops/impl/mean.h"
#include "tensor.h"

#define CUDA_CHECK(err)                                                \
  {                                                                    \
    hipError_t err_ = (err);                                          \
    if (err_ != hipSuccess) {                                         \
      throw std::runtime_error("CUDA Error: " +                        \
                               std::string(hipGetErrorString(err_))); \
    }                                                                  \
  }


Tensor mean_gpu(const Tensor& a, int dim, bool keepdim) {
  // TODO: Sum gpu implemenation
}

