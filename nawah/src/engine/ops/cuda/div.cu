#include <hip/hip_runtime.h>

#include <stdexcept>

#include "allocator/allocatorFactory.h"
#include "device.h"
#include "helpers.h"
#include "engine/ops.h"
#include "tensor.h"

#define CUDA_CHECK(err)                                                \
  {                                                                    \
    hipError_t err_ = (err);                                          \
    if (err_ != hipSuccess) {                                         \
      throw std::runtime_error("CUDA Error: " +                        \
                               std::string(hipGetErrorString(err_))); \
    }                                                                  \
  }


Tensor CudaOps::div(const Tensor& numerator, float denominator) {
  // TODO: div gpu implemenation
}


Tensor CudaOps::div(const Tensor& numerator, const Tensor& denominator) {
  // TODO: div gpu implemenation
}
