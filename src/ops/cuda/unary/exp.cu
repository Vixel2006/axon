#include "hip/hip_runtime.h"
#include "ops/cuda/init.h" // For smalloc, gmalloc (if needed)
#include "ops/cuda/unary.h"

#include "utils/indexing.cuh"

__global__ void noncontig_exp_kernel(const float* a, float* b, int n, const int* a_shape,
                                     const int* a_strides, int a_ndim)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = idx; i < n; i += stride)
    {
        int a_idx = get_idx(a_shape, a_strides, a_ndim, i);
        b[i] = expf(a[a_idx]);
    }
}

__global__ void contig_exp_kernel(const float* a, float* b, int n)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = idx; i < n; i += stride)
    {
        b[i] = expf(a[i]);
    }
}

extern "C" void exp_op_cuda(Tensor* in, Tensor* out)
{
    LOG_INFO("exp_op_cuda: Entering function");
    int N = numel(in->shape, in->ndim);

    int num_threads_per_block = 256;
    int num_blocks = (N + num_threads_per_block - 1) / num_threads_per_block;

    out->data = (Storage*) malloc(sizeof(Storage));
    if (!out->data)
    {
        LOG_ERROR("Failed to allocate Storage for out tensor in exp_op_cuda");
        assert(0 && "Failed to allocate Storage for out tensor in exp_op_cuda");
    }
    out->data->counter = 1;
    out->data->size = N;

    hipError_t err = hipMalloc((void**) &out->data->data, out->data->size * sizeof(float));
    if (err != hipSuccess)
    {
        LOG_ERROR("Failed to allocate CUDA memory for out->data->data in exp_op_cuda: %s",
                  hipGetErrorString(err));
        SAFE_FREE(&out->data, free);
        assert(0 && "Failed to allocate CUDA memory for out->data->data in exp_op_cuda");
    }

    if (is_contiguous(in))
    {
        contig_exp_kernel<<<num_blocks, num_threads_per_block>>>(in->data->data, out->data->data,
                                                                 N);
    }
    else
    {
        noncontig_exp_kernel<<<num_blocks, num_threads_per_block>>>(
            in->data->data, out->data->data, N, in->shape, in->strides, in->ndim);
    }

    CHECK_CUDA();
}
