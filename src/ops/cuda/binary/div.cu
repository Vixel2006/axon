#include "hip/hip_runtime.h"
#include "ops/cuda/binary.h"
#include "ops/cuda/init.h" // For smalloc, gmalloc (if needed)

__global__ void div_kernel(const float* a, const float* b, float* out, const int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = idx; i < n; i += stride)
    {
        out[i] = a[i] / (b[i] + 1e-7f);
    }
}

extern "C" void div_op_cuda(Tensor* a, Tensor* b, Tensor* out)
{
    LOG_INFO("div_op_cuda: Entering function");
    LOG_INFO("Div kernel starts");
    int N = numel(a->shape, a->ndim);

    int num_threads_per_block = 256;
    int num_blocks = (N + num_threads_per_block - 1) / num_threads_per_block;

    out->data = (Storage*) malloc(sizeof(Storage));
    if (!out->data)
    {
        LOG_ERROR("Failed to allocate Storage for out tensor in div_op_cuda");
        assert(0 && "Failed to allocate Storage for out tensor in div_op_cuda");
    }
    out->data->counter = 1;
    out->data->size = N;
    hipError_t err = hipMalloc((void**) &out->data->data, out->data->size * sizeof(float));
    if (err != hipSuccess)
    {
        LOG_ERROR("Failed to allocate CUDA memory for out->data->data in div_op_cuda: %s",
                  hipGetErrorString(err));
        SAFE_FREE(&out->data, free);
        assert(0 && "Failed to allocate CUDA memory for out->data->data in div_op_cuda");
    }

    div_kernel<<<num_blocks, num_threads_per_block>>>(a->data->data, b->data->data, out->data->data,
                                                      N);

    CHECK_CUDA();

    LOG_INFO("Div kernel done successfully");
}
