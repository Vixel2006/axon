#include "hip/hip_runtime.h"
#include "ops/cuda/binary.h"
#include "ops/cuda/init.h"
#include "utils/indexing.cuh"

__global__ void copy_non_contiguous_to_contiguous_kernel(const float* in_data, float* out_data,
                                                         const int* shape, const int* strides,
                                                         int ndim, int num_elements)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = idx; i < num_elements; i += stride)
    {
        int in_idx = get_idx(shape, strides, ndim, i);
        out_data[i] = in_data[in_idx];
    }
}

__global__ void matmul_kernel(const float* a, const float* b, float* out, const int N, const int M,
                              const int K)
{
    int batch = blockIdx.z;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    __shared__ float a_tile[TILE_DIM][TILE_DIM];
    __shared__ float b_tile[TILE_DIM][TILE_DIM];

    float sum = 0.0f;

    const float* a_batch = a + batch * N * K;
    const float* b_batch = b + batch * K * M;
    float* c_batch = out + batch * N * M;

    // NOTE: Here we did (K + TILE_DIM - 1) / TILE_DIM instead of K / TILE_DIM because in the second
    // case if there is leftover that is not divisable by TILE_DIM it will not be calculated
    for (int t = 0; t < (K + TILE_DIM - 1) / TILE_DIM; ++t)
    {
        int tiledColA = t * TILE_DIM + threadIdx.x;
        int tiledRowB = t * TILE_DIM + threadIdx.y;

        a_tile[threadIdx.y][threadIdx.x] =
            (row < N && tiledColA < K) ? a_batch[row * K + tiledColA] : 0.0f;

        b_tile[threadIdx.y][threadIdx.x] =
            (tiledRowB < K && col < M) ? b_batch[tiledRowB * M + col] : 0.0f;
        __syncthreads();

        for (int k = 0; k < TILE_DIM; ++k)
        {
            sum += a_tile[threadIdx.y][k] * b_tile[k][threadIdx.x];
        }

        __syncthreads();
    }

    if (col < M && row < N) c_batch[row * M + col] = sum;
}

extern "C" void matmul_op_cuda(Tensor* a, Tensor* b, Tensor* out, int N, int K, int P)
{
    LOG_INFO("matmul_op_cuda: Entering function with N=%d, K=%d, P=%d", N, K, P);
    LOG_INFO("MATMUL kernel starts......");

    int B = 1;
    for (int i = 0; i < a->ndim - 2; ++i)
    {
        B *= a->shape[i];
    }

    out->data = (Storage*) malloc(sizeof(Storage));
    if (!out->data)
    {
        LOG_ERROR("Failed to allocate Storage for out tensor in matmul_op_cuda");
        assert(0 && "Failed to allocate Storage for out tensor in matmul_op_cuda");
    }
    out->data->counter = 1;
    out->data->size = B * N * P;
    hipError_t err = hipMalloc((void**) &out->data->data, out->data->size * sizeof(float));
    if (err != hipSuccess)
    {
        LOG_ERROR("Failed to allocate CUDA memory for out->data->data in matmul_op_cuda: %s",
                  hipGetErrorString(err));
        SAFE_FREE(&out->data, free);
        assert(0 && "Failed to allocate CUDA memory for out->data->data in matmul_op_cuda");
    }

    float* a_data_ptr = a->data->data;
    float* b_data_ptr = b->data->data;
    float* a_temp_data = NULL;
    float* b_temp_data = NULL;

    int num_elements_a = numel(a->shape, a->ndim);
    int num_elements_b = numel(b->shape, b->ndim);

    int num_threads_per_block = 256;
    int num_blocks_a = (num_elements_a + num_threads_per_block - 1) / num_threads_per_block;
    int num_blocks_b = (num_elements_b + num_threads_per_block - 1) / num_threads_per_block;

    if (!is_contiguous(a))
    {
        hipMalloc((void**) &a_temp_data, num_elements_a * sizeof(float));
        copy_non_contiguous_to_contiguous_kernel<<<num_blocks_a, num_threads_per_block>>>(
            a->data->data, a_temp_data, a->shape, a->strides, a->ndim, num_elements_a);
        a_data_ptr = a_temp_data;
        CHECK_CUDA();
    }

    if (!is_contiguous(b))
    {
        hipMalloc((void**) &b_temp_data, num_elements_b * sizeof(float));
        copy_non_contiguous_to_contiguous_kernel<<<num_blocks_b, num_threads_per_block>>>(
            b->data->data, b_temp_data, b->shape, b->strides, b->ndim, num_elements_b);
        b_data_ptr = b_temp_data;
        CHECK_CUDA();
    }

    dim3 block(TILE_DIM, TILE_DIM);
    dim3 grid((P + TILE_DIM - 1) / TILE_DIM, (N + TILE_DIM - 1) / TILE_DIM, B);

    matmul_kernel<<<grid, block>>>(a_data_ptr, b_data_ptr, out->data->data, N, P, K);

    CHECK_CUDA();

    if (a_temp_data)
    {
        hipFree(a_temp_data);
    }
    if (b_temp_data)
    {
        hipFree(b_temp_data);
    }

    LOG_INFO("MATMUL kernel done successfully");
}
