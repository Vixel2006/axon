#include "ops/cuda/init.h" // For smalloc, gmalloc (if needed)
#include "ops/cuda/reduction.h"

extern "C" void mean_full_op_cuda(Tensor* a, Tensor* out)
{
    LOG_INFO("mean operation on cuda starting......");
    int N = numel(a->shape, a->ndim);

    int num_threads_per_block = 256;
    int blocks = (N + (num_threads_per_block * 2) - 1) / (num_threads_per_block * 2);

    float* h_out_partial_sums;
    float* d_out_partial_sums;

    hipHostMalloc((void**) &h_out_partial_sums, sizeof(float) * blocks);
    hipMalloc((void**) &d_out_partial_sums, sizeof(float) * blocks);

    full_sum_kernel<256><<<blocks, num_threads_per_block, num_threads_per_block * sizeof(float)>>>(
        a->data->data, d_out_partial_sums, N);

    hipMemcpy(h_out_partial_sums, d_out_partial_sums, sizeof(float) * blocks,
               hipMemcpyDeviceToHost);

    float total_sum = 0.0f;
    for (int i = 0; i < blocks; ++i)
    {
        total_sum += h_out_partial_sums[i];
    }

    out->data = (Storage*) malloc(sizeof(Storage));
    if (!out->data)
    {
        LOG_ERROR("Failed to allocate Storage for out tensor in mean_full_op_cuda");
        SAFE_FREE(&h_out_partial_sums, hipHostFree);
        SAFE_FREE(&d_out_partial_sums, hipFree);
        return;
    }
    out->data->counter = 1;
    out->data->size = 1; // Scalar output

    hipError_t err = hipMalloc((void**) &out->data->data, out->data->size * sizeof(float));
    if (err != hipSuccess)
    {
        LOG_ERROR("Failed to allocate CUDA memory for out->data->data in mean_full_op_cuda: %s",
                  hipGetErrorString(err));
        SAFE_FREE(&out->data, free);
        SAFE_FREE(&h_out_partial_sums, hipHostFree);
        SAFE_FREE(&d_out_partial_sums, hipFree);
        return;
    }
    float final_mean = total_sum / N;
    err = hipMemcpy(out->data->data, &final_mean, sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        LOG_ERROR("Failed to copy final mean to CUDA device in mean_full_op_cuda: %s",
                  hipGetErrorString(err));
        SAFE_FREE(&out->data->data, hipFree);
        SAFE_FREE(&out->data, free);
        SAFE_FREE(&h_out_partial_sums, hipHostFree);
        SAFE_FREE(&d_out_partial_sums, hipFree);
        return;
    }

    SAFE_FREE(&h_out_partial_sums, hipHostFree);
    SAFE_FREE(&d_out_partial_sums, hipFree);

    LOG_INFO("mean operation on cuda done.");
}
