#include "hip/hip_runtime.h"
#include "logger.h"
#include "ops/reduction_ops.h"
#include <hip/hip_runtime.h>
#include <float.h>
#include <math.h>

#define CHECK_CUDA()                                                                               \
    do                                                                                             \
    {                                                                                              \
        hipError_t err = hipGetLastError();                                                      \
        if (err != hipSuccess)                                                                    \
        {                                                                                          \
            LOG_ERROR("CUDA runtime error at %s:%d: %s", __FILE__, __LINE__,                       \
                      hipGetErrorString(err));                                                    \
            return;                                                                                \
        }                                                                                          \
    } while (0)

template <int block_size>
__device__ __forceinline__ void wrap_sum_reduce(volatile float* rdata, int tid)
{
    if (block_size >= 64) rdata[tid] += rdata[tid + 32];
    if (block_size >= 32) rdata[tid] += rdata[tid + 16];
    if (block_size >= 16) rdata[tid] += rdata[tid + 8];
    if (block_size >= 8) rdata[tid] += rdata[tid + 4];
    if (block_size >= 4) rdata[tid] += rdata[tid + 2];
    if (block_size >= 2) rdata[tid] += rdata[tid + 1];
}

template <int block_size>
__global__ void sum_kernel(float* a, float* out, int n, int axis_dim, int outer_dim, int inner_dim)
{
    extern __shared__ float rdata[];
    int tid = threadIdx.x;
    int outer_idx = blockIdx.x;
    int inner_idx = blockIdx.y;

    rdata[tid] = 0.0f;
    __syncthreads();

    for (int i = tid; i < axis_dim; i += block_size)
    {
        int current_index = outer_idx * (axis_dim * inner_dim) + i * inner_dim + inner_idx;
        if (current_index < n)
        {
            rdata[tid] += a[current_index];
        }
    }
    __syncthreads();

    if (block_size >= 512)
    {
        if (tid < 256)
        {
            rdata[tid] += rdata[tid + 256];
        }
        __syncthreads();
    }

    if (block_size >= 256)
    {
        if (tid < 128)
        {
            rdata[tid] += rdata[tid + 128];
        }
        __syncthreads();
    }

    if (block_size >= 128)
    {
        if (tid < 64)
        {
            rdata[tid] += rdata[tid + 64];
        }
        __syncthreads();
    }

    if (block_size >= 64) wrap_sum_reduce<block_size>(rdata, tid);

    if (tid == 0)
    {
        int output_idx = outer_idx * inner_dim + inner_idx;
        out[output_idx] = rdata[0];
    }
}

template <int block_size>
__global__ void mean_kernel(float* a, float* out, int n, int axis_dim, int outer_dim, int inner_dim)
{
    extern __shared__ float rdata[];
    int tid = threadIdx.x;
    int outer_idx = blockIdx.x;
    int inner_idx = blockIdx.y;

    rdata[tid] = 0.0f;
    __syncthreads();

    for (int i = tid; i < axis_dim; i += 2 * block_size)
    {
        int left_index = outer_idx * axis_dim * inner_dim + i * inner_dim + inner_idx;
        int right_index =
            outer_idx * axis_dim * inner_dim + (i + block_size) * inner_dim + inner_idx;
        float left_val = left_index < n ? a[left_index] : 0.0f;
        float right_val = right_index < n ? a[right_index] : 0.0f;
        rdata[tid] += left_val + right_val;
    }
    __syncthreads();

    if (block_size >= 512)
    {
        if (tid < 256)
        {
            rdata[tid] += rdata[tid + 256];
        }
        __syncthreads();
    }

    if (block_size >= 256)
    {
        if (tid < 128)
        {
            rdata[tid] += rdata[tid + 128];
        }
        __syncthreads();
    }

    if (block_size >= 128)
    {
        if (tid < 64)
        {
            rdata[tid] += rdata[tid + 64];
        }
        __syncthreads();
    }

    if (block_size >= 64) wrap_sum_reduce<block_size>(rdata, tid);

    if (tid == 0)
    {
        int output_idx = outer_idx * inner_dim + inner_idx;
        out[output_idx] = rdata[0] / axis_dim;
    }
}

template <int block_size>
__device__ __forceinline__ void wrap_max_reduce(volatile float* rdata, int tid)
{
    if (block_size >= 64) rdata[tid] = fmaxf(rdata[tid], rdata[tid + 32]);
    if (block_size >= 32) rdata[tid] = fmaxf(rdata[tid], rdata[tid + 16]);
    if (block_size >= 16) rdata[tid] = fmaxf(rdata[tid], rdata[tid + 8]);
    if (block_size >= 8) rdata[tid] = fmaxf(rdata[tid], rdata[tid + 4]);
    if (block_size >= 4) rdata[tid] = fmaxf(rdata[tid], rdata[tid + 2]);
    if (block_size >= 2) rdata[tid] = fmaxf(rdata[tid], rdata[tid + 1]);
}

template <int block_size>
__global__ void max_kernel(float* a, float* out, int n, int axis_dim, int inner_dim, int outer_dim)
{
    extern __shared__ float rdata[];
    int tid = threadIdx.x;
    int outer_idx = blockIdx.x;
    int inner_idx = blockIdx.y;

    rdata[tid] = -FLT_MAX;
    __syncthreads();

    for (int i = tid; i < axis_dim; i += 2 * block_size)
    {
        int left_index = outer_idx * axis_dim * inner_dim + i * inner_dim + inner_idx;
        int right_index =
            outer_idx * axis_dim * inner_dim + (i + block_size) * inner_dim + inner_idx;
        float left_val = left_index < n ? a[left_index] : -FLT_MAX;
        float right_val = right_index < n ? a[right_index] : -FLT_MAX;
        rdata[tid] = fmaxf(rdata[tid], fmaxf(left_val, right_val));
    }

    if (block_size >= 512)
    {
        if (tid < 256)
        {
            rdata[tid] = fmaxf(rdata[tid], rdata[tid + 256]);
        }
    }

    if (block_size >= 256)
    {
        if (tid < 128)
        {
            rdata[tid] = fmaxf(rdata[tid], rdata[tid + 128]);
        }
    }

    if (block_size >= 128)
    {
        if (tid < 64)
        {
            rdata[tid] = fmaxf(rdata[tid], rdata[tid + 64]);
        }
    }

    if (block_size >= 64) wrap_max_reduce<block_size>(rdata, tid);

    if (tid == 0)
    {
        int output_idx = outer_idx * inner_dim + inner_idx;
        out[output_idx] = fmaxf(out[output_idx], rdata[0]);
    }
}

template <int block_size> __global__ void full_sum_kernel(const float* a, float* out, int n)
{
    extern __shared__ float rdata[];
    int tid = threadIdx.x;
    int idx = blockIdx.x * (block_size * 2) + threadIdx.x;
    int grid_size = blockDim.x * 2 * gridDim.x;
    rdata[tid] = 0;

    while (idx < n)
    {
        rdata[tid] += a[idx] + a[idx + block_size];
        idx += grid_size;
    }
    __syncthreads();

    if (block_size >= 512)
    {
        if (tid < 256)
        {
            rdata[tid] += rdata[tid + 256];
        }
        __syncthreads();
    }

    if (block_size >= 256)
    {
        if (tid < 128)
        {
            rdata[tid] += rdata[tid + 128];
        }
        __syncthreads();
    }

    if (block_size >= 128)
    {
        if (tid < 64)
        {
            rdata[tid] += rdata[tid + 64];
        }
        __syncthreads();
    }

    if (block_size >= 64) wrap_sum_reduce<block_size>(rdata, tid);

    if (tid == 0) out[blockIdx.x] = rdata[0];
}

template <int block_size> __global__ void full_max_kernel(float* a, float* out, int n)
{
    extern __shared__ float rdata[];
    int tid = threadIdx.x;
    int idx = blockIdx.x * (block_size * 2) + threadIdx.x;
    int grid_size = blockDim.x * 2 * gridDim.x;
    rdata[tid] = -FLT_MAX;
    __syncthreads();

    while (idx < n)
    {
        float val1 = a[idx];
        float val2 = (idx + block_size < n) ? a[idx + block_size] : -FLT_MAX;
        rdata[tid] = fmaxf(rdata[tid], fmaxf(val1, val2));
        idx += grid_size;
    }
    __syncthreads();

    if (block_size >= 512)
    {
        if (tid < 256)
        {
            rdata[tid] = fmaxf(rdata[tid], rdata[tid + 256]);
        }
        __syncthreads();
    }

    if (block_size >= 256)
    {
        if (tid < 128)
        {
            rdata[tid] = fmaxf(rdata[tid], rdata[tid + 128]);
        }
        __syncthreads();
    }

    if (block_size >= 128)
    {
        if (tid < 64)
        {
            rdata[tid] = fmaxf(rdata[tid], rdata[tid + 64]);
        }
        __syncthreads();
    }

    if (block_size >= 64) wrap_max_reduce<block_size>(rdata, tid);

    if (tid == 0) out[blockIdx.x] = rdata[0];
}

void sum_op_cuda(Tensor* a, Tensor* out, int axis, bool keepdim)
{
    LOG_INFO("Sum operation on cuda starting......");

    int N = numel(a->shape, a->ndim);

    if (axis < 0 || axis >= a->ndim)
    {
        LOG_ERROR("sum_op_cuda: Axis %d is out of bounds for tensor with %d dimensions.", axis,
                  a->ndim);
        return;
    }

    int outer_dim = 1;
    for (int i = 0; i < axis; ++i)
    {
        outer_dim *= a->shape[i];
    }

    int axis_dim = a->shape[axis];

    int inner_dim = 1;
    for (int i = axis + 1; i < a->ndim; ++i)
    {
        inner_dim *= a->shape[i];
    }

    int num_threads_per_block = 256;
    dim3 grid_dims(outer_dim, inner_dim);

    int output_numel = outer_dim * inner_dim;

    out->data = (Storage*) malloc(sizeof(Storage));
    if (!out->data)
    {
        LOG_ERROR("Failed to allocate Storage for out tensor in sum_op_cuda");
        return;
    }
    out->data->counter = 1;
    out->data->size = output_numel;

    hipError_t err = hipMalloc((void**) &out->data->data, out->data->size * sizeof(float));
    if (err != hipSuccess)
    {
        LOG_ERROR("Failed to allocate CUDA memory for out->data->data in sum_op_cuda: %s",
                  hipGetErrorString(err));
        SAFE_FREE(&out->data, free);
        return;
    }

    sum_kernel<256><<<grid_dims, num_threads_per_block, num_threads_per_block * sizeof(float)>>>(
        a->data->data, out->data->data, N, axis_dim, outer_dim, inner_dim);

    CHECK_CUDA();
    LOG_INFO("Sum operation on cuda done.");
}

void mean_op_cuda(Tensor* a, Tensor* out, int axis, bool keepdim)
{
    LOG_INFO("mean operation on cuda starting......");

    int N = numel(a->shape, a->ndim);

    if (axis < 0 || axis >= a->ndim)
    {
        LOG_ERROR("mean_op_cuda: Axis %d is out of bounds for tensor with %d dimensions.", axis,
                  a->ndim);
        return;
    }

    int outer_dim = 1;
    for (int i = 0; i < axis; ++i)
    {
        outer_dim *= a->shape[i];
    }

    int axis_dim = a->shape[axis];

    int inner_dim = 1;
    for (int i = axis + 1; i < a->ndim; ++i)
    {
        inner_dim *= a->shape[i];
    }

    int num_threads_per_block = 256;
    dim3 grid_dims(outer_dim, inner_dim);

    int output_numel = outer_dim * inner_dim;

    out->data = (Storage*) malloc(sizeof(Storage));
    if (!out->data)
    {
        LOG_ERROR("Failed to allocate Storage for out tensor in mean_op_cuda");
        return;
    }
    out->data->counter = 1;
    out->data->size = output_numel;

    hipError_t err = hipMalloc((void**) &out->data->data, out->data->size * sizeof(float));
    if (err != hipSuccess)
    {
        LOG_ERROR("Failed to allocate CUDA memory for out->data->data in mean_op_cuda: %s",
                  hipGetErrorString(err));
        SAFE_FREE(&out->data, free);
        return;
    }

    mean_kernel<256><<<grid_dims, num_threads_per_block, num_threads_per_block * sizeof(float)>>>(
        a->data->data, out->data->data, N, axis_dim, outer_dim, inner_dim);

    CHECK_CUDA();
    LOG_INFO("mean operation on cuda done.");
}

void max_op_cuda(Tensor* a, Tensor* out, int axis, bool keepdim)
{
    LOG_INFO("max operation on cuda starting......");

    int N = numel(a->shape, a->ndim);

    if (axis < 0 || axis >= a->ndim)
    {
        LOG_ERROR("max_op_cuda: Axis %d is out of bounds for tensor with %d dimensions.", axis,
                  a->ndim);
        return;
    }

    int outer_dim = 1;
    for (int i = 0; i < axis; ++i)
    {
        outer_dim *= a->shape[i];
    }

    int axis_dim = a->shape[axis];

    int inner_dim = 1;
    for (int i = axis + 1; i < a->ndim; ++i)
    {
        inner_dim *= a->shape[i];
    }

    int num_threads_per_block = 256;
    dim3 grid_dims(outer_dim, inner_dim);

    int output_numel = outer_dim * inner_dim;

    out->data = (Storage*) malloc(sizeof(Storage));
    if (!out->data)
    {
        LOG_ERROR("Failed to allocate Storage for out tensor in max_op_cuda");
        return;
    }
    out->data->counter = 1;
    out->data->size = output_numel;

    hipError_t err = hipMalloc((void**) &out->data->data, out->data->size * sizeof(float));
    if (err != hipSuccess)
    {
        LOG_ERROR("Failed to allocate CUDA memory for out->data->data in max_op_cuda: %s",
                  hipGetErrorString(err));
        SAFE_FREE(&out->data, free);
        return;
    }

    max_kernel<256><<<grid_dims, num_threads_per_block, num_threads_per_block * sizeof(float)>>>(
        a->data->data, out->data->data, N, axis_dim, inner_dim, outer_dim);

    CHECK_CUDA();
    LOG_INFO("max operation on cuda done.");
}

void sum_full_op_cuda(Tensor* a, Tensor* out)
{
    LOG_INFO("Sum operation on cuda starting......");
    int N = numel(a->shape, a->ndim);

    int num_threads_per_block = 256;
    int blocks = (N + (num_threads_per_block * 2) - 1) / (num_threads_per_block * 2);

    float* h_out_partial_sums;
    float* d_out_partial_sums;

    hipHostMalloc((void**) &h_out_partial_sums, sizeof(float) * blocks);
    hipMalloc((void**) &d_out_partial_sums, sizeof(float) * blocks);

    full_sum_kernel<256><<<blocks, num_threads_per_block, num_threads_per_block * sizeof(float)>>>(
        a->data->data, d_out_partial_sums, N);

    hipMemcpy(h_out_partial_sums, d_out_partial_sums, sizeof(float) * blocks,
               hipMemcpyDeviceToHost);

    float total_sum = 0.0f;
    for (int i = 0; i < blocks; ++i)
    {
        total_sum += h_out_partial_sums[i];
    }

    out->data = (Storage*) malloc(sizeof(Storage));
    if (!out->data)
    {
        LOG_ERROR("Failed to allocate Storage for out tensor in sum_full_op_cuda");
        SAFE_FREE(&h_out_partial_sums, hipHostFree);
        SAFE_FREE(&d_out_partial_sums, hipFree);
        return;
    }
    out->data->counter = 1;
    out->data->size = 1; // Scalar output

    hipError_t err = hipMalloc((void**) &out->data->data, out->data->size * sizeof(float));
    if (err != hipSuccess)
    {
        LOG_ERROR("Failed to allocate CUDA memory for out->data->data in sum_full_op_cuda: %s",
                  hipGetErrorString(err));
        SAFE_FREE(&out->data, free);
        SAFE_FREE(&h_out_partial_sums, hipHostFree);
        SAFE_FREE(&d_out_partial_sums, hipFree);
        return;
    }
    err = hipMemcpy(out->data->data, &total_sum, sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        LOG_ERROR("Failed to copy final sum to CUDA device in sum_full_op_cuda: %s",
                  hipGetErrorString(err));
        SAFE_FREE(&out->data->data, hipFree);
        SAFE_FREE(&out->data, free);
        SAFE_FREE(&h_out_partial_sums, hipHostFree);
        SAFE_FREE(&d_out_partial_sums, hipFree);
        return;
    }

    SAFE_FREE(&h_out_partial_sums, hipHostFree);
    SAFE_FREE(&d_out_partial_sums, hipFree);

    LOG_INFO("Sum operation on cuda done.");
}

void mean_full_op_cuda(Tensor* a, Tensor* out)
{
    LOG_INFO("mean operation on cuda starting......");
    int N = numel(a->shape, a->ndim);

    int num_threads_per_block = 256;
    int blocks = (N + (num_threads_per_block * 2) - 1) / (num_threads_per_block * 2);

    float* h_out_partial_sums;
    float* d_out_partial_sums;

    hipHostMalloc((void**) &h_out_partial_sums, sizeof(float) * blocks);
    hipMalloc((void**) &d_out_partial_sums, sizeof(float) * blocks);

    full_sum_kernel<256><<<blocks, num_threads_per_block, num_threads_per_block * sizeof(float)>>>(
        a->data->data, d_out_partial_sums, N);

    hipMemcpy(h_out_partial_sums, d_out_partial_sums, sizeof(float) * blocks,
               hipMemcpyDeviceToHost);

    float total_sum = 0.0f;
    for (int i = 0; i < blocks; ++i)
    {
        total_sum += h_out_partial_sums[i];
    }

    out->data = (Storage*) malloc(sizeof(Storage));
    if (!out->data)
    {
        LOG_ERROR("Failed to allocate Storage for out tensor in mean_full_op_cuda");
        SAFE_FREE(&h_out_partial_sums, hipHostFree);
        SAFE_FREE(&d_out_partial_sums, hipFree);
        return;
    }
    out->data->counter = 1;
    out->data->size = 1; // Scalar output

    hipError_t err = hipMalloc((void**) &out->data->data, out->data->size * sizeof(float));
    if (err != hipSuccess)
    {
        LOG_ERROR("Failed to allocate CUDA memory for out->data->data in mean_full_op_cuda: %s",
                  hipGetErrorString(err));
        SAFE_FREE(&out->data, free);
        SAFE_FREE(&h_out_partial_sums, hipHostFree);
        SAFE_FREE(&d_out_partial_sums, hipFree);
        return;
    }
    float final_mean = total_sum / N;
    err = hipMemcpy(out->data->data, &final_mean, sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        LOG_ERROR("Failed to copy final mean to CUDA device in mean_full_op_cuda: %s",
                  hipGetErrorString(err));
        SAFE_FREE(&out->data->data, hipFree);
        SAFE_FREE(&out->data, free);
        SAFE_FREE(&h_out_partial_sums, hipHostFree);
        SAFE_FREE(&d_out_partial_sums, hipFree);
        return;
    }

    SAFE_FREE(&h_out_partial_sums, hipHostFree);
    SAFE_FREE(&d_out_partial_sums, hipFree);

    LOG_INFO("mean operation on cuda done.");
}

void max_full_op_cuda(Tensor* a, Tensor* out)
{
    LOG_INFO("max operation on cuda starting......");
    int N = numel(a->shape, a->ndim);

    int num_threads_per_block = 256;
    int blocks = (N + (num_threads_per_block * 2) - 1) / (num_threads_per_block * 2);

    float* h_out_partial_maxs;
    float* d_out_partial_maxs;

    hipHostMalloc((void**) &h_out_partial_maxs, sizeof(float) * blocks);
    hipMalloc((void**) &d_out_partial_maxs, sizeof(float) * blocks);

    full_max_kernel<256><<<blocks, num_threads_per_block, num_threads_per_block * sizeof(float)>>>(
        a->data->data, d_out_partial_maxs, N);

    hipMemcpy(h_out_partial_maxs, d_out_partial_maxs, sizeof(float) * blocks,
               hipMemcpyDeviceToHost);

    float max = h_out_partial_maxs[0];
    for (int i = 0; i < blocks; ++i)
    {
        max = fmaxf(max, h_out_partial_maxs[i]);
    }

    out->data = (Storage*) malloc(sizeof(Storage));
    if (!out->data)
    {
        LOG_ERROR("Failed to allocate Storage for out tensor in max_full_op_cuda");
        SAFE_FREE(&h_out_partial_maxs, hipHostFree);
        SAFE_FREE(&d_out_partial_maxs, hipFree);
        return;
    }
    out->data->counter = 1;
    out->data->size = 1; // Scalar output

    hipError_t err = hipMalloc((void**) &out->data->data, out->data->size * sizeof(float));
    if (err != hipSuccess)
    {
        LOG_ERROR("Failed to allocate CUDA memory for out->data->data in max_full_op_cuda: %s",
                  hipGetErrorString(err));
        SAFE_FREE(&out->data, free);
        SAFE_FREE(&h_out_partial_maxs, hipHostFree);
        SAFE_FREE(&d_out_partial_maxs, hipFree);
        return;
    }
    err = hipMemcpy(out->data->data, &max, sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        LOG_ERROR("Failed to copy final max to CUDA device in max_full_op_cuda: %s",
                  hipGetErrorString(err));
        SAFE_FREE(&out->data->data, hipFree);
        SAFE_FREE(&out->data, free);
        SAFE_FREE(&h_out_partial_maxs, hipHostFree);
        SAFE_FREE(&d_out_partial_maxs, hipFree);
        return;
    }

    SAFE_FREE(&h_out_partial_maxs, hipHostFree);
    SAFE_FREE(&d_out_partial_maxs, hipFree);

    LOG_INFO("max operation on cuda done.");
}
