#include "hip/hip_runtime.h"
#include "logger.h"
#include "ops/binary_ops.h"
#include "ops/init_ops.h"
#include <hip/hip_runtime.h>

#define TILE_DIM 16

#define CHECK_CUDA()                                                                               \
    do                                                                                             \
    {                                                                                              \
        hipError_t err = hipGetLastError();                                                      \
        if (err != hipSuccess)                                                                    \
        {                                                                                          \
            LOG_ERROR("CUDA runtime error at %s:%d: %s", __FILE__, __LINE__,                       \
                      hipGetErrorString(err));                                                    \
            return;                                                                                \
        }                                                                                          \
    } while (0)

__global__ void add_kernel(const float* a, const float* b, float* out, const int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = idx; i < n; i += stride)
    {
        out[i] = a[i] + b[i];
    }
}

__global__ void sub_kernel(const float* a, const float* b, float* out, const int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = idx; i < n; i += stride)
    {
        out[i] = a[i] - b[i];
    }
}

__global__ void mul_kernel(const float* a, const float* b, float* out, const int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = idx; i < n; i += stride)
    {
        out[i] = a[i] * b[i];
    }
}

__global__ void div_kernel(const float* a, const float* b, float* out, const int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = idx; i < n; i += stride)
    {
        out[i] = a[i] / b[i];
    }
}

__global__ void pow_kernel(const float* a, const float* b, float* out, const int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = idx; i < n; i += stride)
    {
        out[i] = powf(a[i], b[i]);
    }
}

__global__ void matmul_kernel(const float* a, const float* b, float* out, const int N, const int M,
                              const int K)
{
    int batch = blockIdx.z;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    __shared__ float a_tile[TILE_DIM][TILE_DIM];
    __shared__ float b_tile[TILE_DIM][TILE_DIM];

    float sum = 0.0f;

    const float* a_batch = a + batch * N * K;
    const float* b_batch = b + batch * K * M;
    float* c_batch = out + batch * N * M;

    for (int t = 0; t < (K + TILE_DIM - 1) / TILE_DIM; ++t) // Corrected outer loop
    {
        int tiledColA = t * TILE_DIM + threadIdx.x;
        int tiledRowB = t * TILE_DIM + threadIdx.y;

        a_tile[threadIdx.y][threadIdx.x] =
            (row < N && tiledColA < K) ? a_batch[row * K + tiledColA] : 0.0f;

        b_tile[threadIdx.y][threadIdx.x] =
            (tiledRowB < K && col < M) ? b_batch[tiledRowB * M + col] : 0.0f;
        __syncthreads();

        for (int k = 0; k < TILE_DIM; ++k) // Corrected inner loop
        {
            sum += a_tile[threadIdx.y][k] * b_tile[k][threadIdx.x];
        }

        __syncthreads();
    }

    if (col < M && row < N) c_batch[row * M + col] = sum;
}

void add_op_cuda(Tensor* a, Tensor* b, Tensor* out)
{
    LOG_INFO("Add kernel starts");
    int N = numel(a->shape, a->ndim);

    int num_threads_per_block = 256;
    int num_blocks = (N + num_threads_per_block - 1) / num_threads_per_block;

    float* h_out;
    float* d_out;

    hipHostMalloc((void**) &h_out, sizeof(float) * N);
    hipMalloc((void**) &d_out, sizeof(float) * N);

    add_kernel<<<num_blocks, num_threads_per_block>>>(a->data->data, b->data->data, d_out, N);

    hipMemcpy(h_out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
    from_data(out, h_out);

    SAFE_FREE(&d_out, hipFree);
    SAFE_FREE(&h_out, hipHostFree);

    CHECK_CUDA();

    LOG_INFO("Add kernel done successfully");
}

void sub_op_cuda(Tensor* a, Tensor* b, Tensor* out)
{
    LOG_INFO("Sub kernel starts");
    int N = numel(a->shape, a->ndim);

    int num_threads_per_block = 256;
    int num_blocks = (N + num_threads_per_block - 1) / num_threads_per_block;

    float* h_out;
    float* d_out;

    hipHostMalloc((void**) &h_out, sizeof(float) * N);
    hipMalloc((void**) &d_out, sizeof(float) * N);

    sub_kernel<<<num_blocks, num_threads_per_block>>>(a->data->data, b->data->data, d_out, N);
    hipMemcpy(h_out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
    from_data(out, h_out);

    SAFE_FREE(&d_out, hipFree);
    SAFE_FREE(&h_out, hipHostFree);

    CHECK_CUDA();

    LOG_INFO("Sub kernel done successfully");
}

void mul_op_cuda(Tensor* a, Tensor* b, Tensor* out)
{
    LOG_INFO("Mul kernel starts");
    int N = numel(a->shape, a->ndim);

    int num_threads_per_block = 256;
    int num_blocks = (N + num_threads_per_block - 1) / num_threads_per_block;

    float* h_out;
    float* d_out;

    hipHostMalloc((void**) &h_out, sizeof(float) * N);
    hipMalloc((void**) &d_out, sizeof(float) * N);

    mul_kernel<<<num_blocks, num_threads_per_block>>>(a->data->data, b->data->data, d_out, N);

    hipMemcpy(h_out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
    from_data(out, h_out);

    SAFE_FREE(&d_out, hipFree);
    SAFE_FREE(&h_out, hipHostFree);
    CHECK_CUDA();

    LOG_INFO("Mul kernel done successfully");
}

void div_op_cuda(Tensor* a, Tensor* b, Tensor* out)
{
    LOG_INFO("Div kernel starts");
    int N = numel(a->shape, a->ndim);

    int num_threads_per_block = 256;
    int num_blocks = (N + num_threads_per_block - 1) / num_threads_per_block;

    float* h_out;
    float* d_out;

    hipHostMalloc((void**) &h_out, sizeof(float) * N);
    hipMalloc((void**) &d_out, sizeof(float) * N);

    div_kernel<<<num_blocks, num_threads_per_block>>>(a->data->data, b->data->data, d_out, N);

    hipMemcpy(h_out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
    from_data(out, h_out);

    SAFE_FREE(&d_out, hipFree);
    SAFE_FREE(&h_out, hipHostFree);

    CHECK_CUDA();

    LOG_INFO("Div kernel done successfully");
}

void pow_op_cuda(Tensor* a, Tensor* b, Tensor* out)
{
    LOG_INFO("Pow kernel starts......");
    int N = numel(a->shape, a->ndim);

    int num_threads_per_block = 256;
    int num_blocks = (N + num_threads_per_block - 1) / num_threads_per_block;

    float* h_out;
    float* d_out;

    hipHostMalloc((void**) &h_out, sizeof(float) * N);
    hipMalloc((void**) &d_out, sizeof(float) * N);

    pow_kernel<<<num_blocks, num_threads_per_block>>>(a->data->data, b->data->data, d_out, N);

    hipMemcpy(h_out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
    from_data(out, h_out);

    SAFE_FREE(&d_out, hipFree);
    SAFE_FREE(&h_out, hipHostFree);

    CHECK_CUDA();

    LOG_INFO("Pow kernel done successfully");
}

void matmul_op_cuda(Tensor* a, Tensor* b, Tensor* out, int N, int K, int P)
{
    LOG_INFO("MATMUL kernel starts......");

    int B = 1;
    for (int i = 0; i < a->ndim - 2; ++i)
    {
        B += a->shape[i];
    }

    float* h_out;
    float* d_out;

    hipHostMalloc((void**) &h_out, sizeof(float) * B * N * P);
    hipMalloc((void**) &d_out, sizeof(float) * B * N * P);

    dim3 block(TILE_DIM, TILE_DIM);
    dim3 grid((P + TILE_DIM - 1) / TILE_DIM, (N + TILE_DIM - 1) / TILE_DIM, B);

    matmul_kernel<<<grid, block>>>(a->data->data, b->data->data, d_out, N, P, K);

    hipMemcpy(h_out, d_out, sizeof(float) * N * B * P, hipMemcpyDeviceToHost);

    from_data(out, h_out);

    SAFE_FREE(&d_out, hipFree);
    SAFE_FREE(&h_out, hipHostFree);

    CHECK_CUDA();

    LOG_INFO("MATMUL kernel done successfully");
}

void conv2d_op_cuda(Tensor* in, Tensor* kernel, Tensor* out, const int* kernel_size,
                    const int* stride, const int padding)
{
    (void) in;
    (void) kernel;
    (void) out;
    (void) kernel_size;
    (void) stride;
    (void) padding;
    LOG_WARN("conv2d_op_cuda: CUDA implementation not available yet.");
}

void dot_op_cuda(Tensor* a, Tensor* b, Tensor* out)
{
    (void) a;
    (void) b;
    (void) out;
    LOG_WARN("dot_op_cuda: CUDA implementation not available yet.");
}
