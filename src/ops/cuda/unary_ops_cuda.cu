#include "hip/hip_runtime.h"
#include "logger.h"
#include "ops/init_ops.h"
#include "ops/unary_ops.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>

#define CHECK_CUDA()                                                                               \
    do                                                                                             \
    {                                                                                              \
        hipError_t err = hipGetLastError();                                                      \
        if (err != hipSuccess)                                                                    \
        {                                                                                          \
            LOG_ERROR("CUDA runtime error at %s:%d: %s", __FILE__, __LINE__,                       \
                      hipGetErrorString(err));                                                    \
            return;                                                                                \
        }                                                                                          \
    } while (0)

__global__ void relu_kernel(const float* a, float* b, int n)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = idx; i < n; i += stride)
    {
        // NOTE: Here we use the fmaxf from the cuda math library so we can metigate the diversion
        // that can happens in the warp calcuation if we used b[i] = a[i] > 0 ? a[i] : 0;
        b[i] = fmaxf(a[i], 0);
    }
}

__global__ void log_kernel(const float* a, float* b, int n)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = idx; i < n; i += stride)
    {
        b[i] = logf(a[i] + 1e-7f);
    }
}

__global__ void exp_kernel(const float* a, float* b, int n)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = idx; i < n; i += stride)
    {
        b[i] = expf(a[i]);
    }
}

__global__ void neg_kernel(const float* a, float* b, int n)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = idx; i < n; i += stride)
    {
        b[i] = -a[i];
    }
}

__global__ void abs_kernel(const float* a, float* b, int n)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = idx; i < n; i += stride)
    {
        b[i] = fabsf(a[i]);
    }
}

__global__ void clip_kernel(const float* a, float* b, const float min_val, const float max_val,
                            int n)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = idx; i < n; i += stride)
    {
        // NOTE: Here we will use a mix of fminf, fmaxf to metigate the warp diversion problem that
        // will happen if use branching
        b[i] = fmaxf(min_val, fminf(max_val, a[i]));
    }
}

// TODO: There's too many memory copies in this implementation we will need to fix this in the
// future
void relu_op_cuda(Tensor* in, Tensor* out)
{
    LOG_INFO("ReLU operation on CUDA running......");
    int N = numel(in->shape, in->ndim);

    int num_threads_per_block = 256;
    int num_blocks = (N + num_threads_per_block - 1) / num_threads_per_block;

    out->data = (Storage*) malloc(sizeof(Storage));
    if (!out->data)
    {
        LOG_ERROR("Failed to allocate Storage for out tensor in relu_op_cuda");
        return;
    }
    out->data->counter = 1;
    out->data->size = N;

    hipError_t err = hipMalloc((void**) &out->data->data, out->data->size * sizeof(float));
    if (err != hipSuccess)
    {
        LOG_ERROR("Failed to allocate CUDA memory for out->data->data in relu_op_cuda: %s",
                  hipGetErrorString(err));
        SAFE_FREE(&out->data, free);
        return;
    }

    relu_kernel<<<num_blocks, num_threads_per_block>>>(in->data->data, out->data->data, N);

    CHECK_CUDA();

    LOG_INFO("ReLU operation done on CUDA successfully.");
}

void log_op_cuda(Tensor* in, Tensor* out)
{
    LOG_INFO("LOG operation on CUDA running.......");
    int N = numel(in->shape, in->ndim);

    int num_threads_per_block = 256;
    int num_blocks = (N + num_threads_per_block - 1) / num_threads_per_block;

    out->data = (Storage*) malloc(sizeof(Storage));
    if (!out->data)
    {
        LOG_ERROR("Failed to allocate Storage for out tensor in log_op_cuda");
        return;
    }
    out->data->counter = 1;
    out->data->size = N;

    hipError_t err = hipMalloc((void**) &out->data->data, out->data->size * sizeof(float));
    if (err != hipSuccess)
    {
        LOG_ERROR("Failed to allocate CUDA memory for out->data->data in log_op_cuda: %s",
                  hipGetErrorString(err));
        SAFE_FREE(&out->data, free);
        return;
    }

    log_kernel<<<num_blocks, num_threads_per_block>>>(in->data->data, out->data->data, N);

    CHECK_CUDA();

    LOG_INFO("LOG operation done on CUDA successfully.");
}

void exp_op_cuda(Tensor* in, Tensor* out)
{
    LOG_INFO("EXP operation on CUDA running.......");
    int N = numel(in->shape, in->ndim);

    int num_threads_per_block = 256;
    int num_blocks = (N + num_threads_per_block - 1) / num_threads_per_block;

    out->data = (Storage*) malloc(sizeof(Storage));
    if (!out->data)
    {
        LOG_ERROR("Failed to allocate Storage for out tensor in exp_op_cuda");
        return;
    }
    out->data->counter = 1;
    out->data->size = N;

    hipError_t err = hipMalloc((void**) &out->data->data, out->data->size * sizeof(float));
    if (err != hipSuccess)
    {
        LOG_ERROR("Failed to allocate CUDA memory for out->data->data in exp_op_cuda: %s",
                  hipGetErrorString(err));
        SAFE_FREE(&out->data, free);
        return;
    }

    exp_kernel<<<num_blocks, num_threads_per_block>>>(in->data->data, out->data->data, N);

    CHECK_CUDA();

    LOG_INFO("EXP operation done on CUDA successfully.");
}

void neg_op_cuda(Tensor* in, Tensor* out)
{
    LOG_INFO("NEG operation on CUDA running.......");
    int N = numel(in->shape, in->ndim);

    int num_threads_per_block = 256;
    int num_blocks = (N + num_threads_per_block - 1) / num_threads_per_block;

    out->data = (Storage*) malloc(sizeof(Storage));
    if (!out->data)
    {
        LOG_ERROR("Failed to allocate Storage for out tensor in neg_op_cuda");
        return;
    }
    out->data->counter = 1;
    out->data->size = N;

    hipError_t err = hipMalloc((void**) &out->data->data, out->data->size * sizeof(float));
    if (err != hipSuccess)
    {
        LOG_ERROR("Failed to allocate CUDA memory for out->data->data in neg_op_cuda: %s",
                  hipGetErrorString(err));
        SAFE_FREE(&out->data, free);
        return;
    }

    neg_kernel<<<num_blocks, num_threads_per_block>>>(in->data->data, out->data->data, N);

    CHECK_CUDA();

    LOG_INFO("NEG operation done on CUDA successfully.");
}

void abs_op_cuda(Tensor* in, Tensor* out)
{
    LOG_INFO("ABS operation on CUDA running.......");
    int N = numel(in->shape, in->ndim);

    int num_threads_per_block = 256;
    int num_blocks = (N + num_threads_per_block - 1) / num_threads_per_block;

    out->data = (Storage*) malloc(sizeof(Storage));
    if (!out->data)
    {
        LOG_ERROR("Failed to allocate Storage for out tensor in abs_op_cuda");
        return;
    }
    out->data->counter = 1;
    out->data->size = N;

    hipError_t err = hipMalloc((void**) &out->data->data, out->data->size * sizeof(float));
    if (err != hipSuccess)
    {
        LOG_ERROR("Failed to allocate CUDA memory for out->data->data in abs_op_cuda: %s",
                  hipGetErrorString(err));
        SAFE_FREE(&out->data, free);
        return;
    }

    abs_kernel<<<num_blocks, num_threads_per_block>>>(in->data->data, out->data->data, N);

    CHECK_CUDA();

    LOG_INFO("ABS operation done on CUDA successfully.");
}

void clip_op_cuda(Tensor* in, Tensor* out, float min_val, float max_val)
{
    LOG_INFO("CLIP operation on CUDA running.......");
    int N = numel(in->shape, in->ndim);

    int num_threads_per_block = 256;
    int num_blocks = (N + num_threads_per_block - 1) / num_threads_per_block;

    out->data = (Storage*) malloc(sizeof(Storage));
    if (!out->data)
    {
        LOG_ERROR("Failed to allocate Storage for out tensor in clip_op_cuda");
        return;
    }
    out->data->counter = 1;
    out->data->size = N;

    hipError_t err = hipMalloc((void**) &out->data->data, out->data->size * sizeof(float));
    if (err != hipSuccess)
    {
        LOG_ERROR("Failed to allocate CUDA memory for out->data->data in clip_op_cuda: %s",
                  hipGetErrorString(err));
        SAFE_FREE(&out->data, free);
        return;
    }

    clip_kernel<<<num_blocks, num_threads_per_block>>>(in->data->data, out->data->data, min_val,
                                                       max_val, N);

    CHECK_CUDA();

    LOG_INFO("CLIP operation done on CUDA successfully.");
}
