#include "hip/hip_runtime.h"
#include "logger.h"
#include "ops/init_ops.h"
#include "ops/unary_ops.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>

#define CHECK_CUDA()                                                                               \
    do                                                                                             \
    {                                                                                              \
        hipError_t err = hipGetLastError();                                                      \
        if (err != hipSuccess)                                                                    \
        {                                                                                          \
            LOG_ERROR("CUDA runtime error at %s:%d: %s", __FILE__, __LINE__,                       \
                      hipGetErrorString(err));                                                    \
            return;                                                                                \
        }                                                                                          \
    } while (0)

__global__ void relu_kernel(const float* a, float* b, int n)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = idx; i < n; i += stride)
    {
        // NOTE: Here we use the fmaxf from the cuda math library so we can metigate the diversion
        // that can happens in the warp calcuation if we used b[i] = a[i] > 0 ? a[i] : 0;
        b[i] = fmaxf(a[i], 0);
    }
}

__global__ void log_kernel(const float* a, float* b, int n)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = idx; i < n; i += stride)
    {
        b[i] = logf(a[i]);
    }
}

__global__ void exp_kernel(const float* a, float* b, int n)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = idx; i < n; i += stride)
    {
        b[i] = expf(a[i]);
    }
}

__global__ void neg_kernel(const float* a, float* b, int n)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = idx; i < n; i += stride)
    {
        b[i] = -a[i];
    }
}

__global__ void abs_kernel(const float* a, float* b, int n)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = idx; i < n; i += stride)
    {
        b[i] = fabsf(a[i]);
    }
}

__global__ void clip_kernel(const float* a, float* b, const float min_val, const float max_val,
                            int n)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = idx; i < n; i += stride)
    {
        // NOTE: Here we will use a mix of fminf, fmaxf to metigate the warp diversion problem that
        // will happen if use branching
        b[i] = fmaxf(min_val, fminf(max_val, a[i]));
    }
}

void relu_op_cuda(Tensor* in, Tensor* out)
{
    LOG_INFO("ReLU operation on CUDA running......");
    int N = numel(in->shape, in->ndim);

    int num_threads_per_block = 256;
    int num_blocks = (N + num_threads_per_block - 1) / num_threads_per_block;

    float* d_data;
    hipMalloc((void**) &d_data, sizeof(float) * N);

    relu_kernel<<<num_blocks, num_threads_per_block>>>(in->data->data, d_data, N);

    CHECK_CUDA();

    float* h_data;
    hipHostMalloc((void**) &h_data, sizeof(float) * N);
    hipMemcpy(h_data, d_data, N * sizeof(float), hipMemcpyDeviceToHost);

    from_data(out, h_data);

    SAFE_FREE(&h_data, hipHostFree);
    SAFE_FREE(&d_data, hipFree);

    LOG_INFO("ReLU operation done on CUDA successfully.");
}

void log_op_cuda(Tensor* in, Tensor* out)
{
    LOG_INFO("LOG operation on CUDA running.......");
    int N = numel(in->shape, in->ndim);

    int num_threads_per_block = 256;
    int num_blocks = (N + num_threads_per_block - 1) / num_threads_per_block;

    log_kernel<<<num_blocks, num_threads_per_block>>>(in->data->data, out->data->data, N);

    CHECK_CUDA();

    LOG_INFO("LOG operation done on CUDA successfully.");
}

void exp_op_cuda(Tensor* in, Tensor* out)
{
    LOG_INFO("EXP operation on CUDA running.......");
    int N = numel(in->shape, in->ndim);

    int num_threads_per_block = 256;
    int num_blocks = (N + num_threads_per_block - 1) / num_threads_per_block;

    exp_kernel<<<num_blocks, num_threads_per_block>>>(in->data->data, out->data->data, N);

    CHECK_CUDA();

    LOG_INFO("EXP operation done on CUDA successfully.");
}

void neg_op_cuda(Tensor* in, Tensor* out)
{
    LOG_INFO("NEG operation on CUDA running.......");
    int N = numel(in->shape, in->ndim);

    int num_threads_per_block = 256;
    int num_blocks = (N + num_threads_per_block - 1) / num_threads_per_block;

    neg_kernel<<<num_blocks, num_threads_per_block>>>(in->data->data, out->data->data, N);

    CHECK_CUDA();

    LOG_INFO("NEG operation done on CUDA successfully.");
}

void abs_op_cuda(Tensor* in, Tensor* out)
{
    LOG_INFO("ABS operation on CUDA running.......");
    int N = numel(in->shape, in->ndim);

    int num_threads_per_block = 256;
    int num_blocks = (N + num_threads_per_block - 1) / num_threads_per_block;

    abs_kernel<<<num_blocks, num_threads_per_block>>>(in->data->data, out->data->data, N);

    CHECK_CUDA();

    LOG_INFO("ABS operation done on CUDA successfully.");
}

void clip_op_cuda(Tensor* in, Tensor* out, float min_val, float max_val)
{
    LOG_INFO("CLIP operation on CUDA running.......");
    int N = numel(in->shape, in->ndim);

    int num_threads_per_block = 256;
    int num_blocks = (N + num_threads_per_block - 1) / num_threads_per_block;

    clip_kernel<<<num_blocks, num_threads_per_block>>>(in->data->data, out->data->data, min_val,
                                                       max_val, N);

    CHECK_CUDA();

    LOG_INFO("CLIP operation done on CUDA successfully.");
}
