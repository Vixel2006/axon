#include "hip/hip_runtime.h"
#include "engine/ops.h"
#include "tensor.h"
#include "helpers.h"
#include "allocator/allocatorFactory.h"
#include <hip/hip_runtime.h>
#include <stdexcept>

#define CUDA_CHECK(call)                                                    \
    do {                                                                    \
        hipError_t err = call;                                             \
        if (err != hipSuccess) {                                           \
            throw std::runtime_error(std::string("CUDA Error in " #call " : ") + \
                                     hipGetErrorString(err));              \
        }                                                                   \
    } while (0)

__global__ void exp_kernel(const float* a_data, float* c_data, size_t num_elements) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = index; i < num_elements; i += stride) {
        c_data[i] = expf(a_data[i]); // Use expf for single-precision floats
    }
}

Tensor CudaOps::exp(const Tensor &a) {
    if (a.device().type != DeviceType::CUDA) {
        throw std::runtime_error("Input tensor for CudaOps::exp must be on the CUDA device.");
    }

    const size_t num_elements = a.numel();
    if (num_elements == 0) {
        return Tensor(a.shape(), a.dtype(), deviceToString(a.device()), false);
    }
    const size_t data_size = num_elements * sizeof(float);

    const float* d_a = static_cast<const float*>(a.raw_ptr());

    auto allocator = AllocatorFactory::get(a.device());
    void* d_c_raw = allocator->allocate(data_size);
    if (!d_c_raw) {
        throw std::runtime_error("Failed to allocate CUDA memory for output tensor via AllocatorFactory.");
    }
    float* d_c = static_cast<float*>(d_c_raw);

    const int threadsPerBlock = 256;
    const int blocksPerGrid = (num_elements + threadsPerBlock - 1) / threadsPerBlock;

    exp_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_c, num_elements);
    CUDA_CHECK(hipGetLastError());

    auto deleter = [allocator](void *ptr) { allocator->deallocate(ptr); };
    std::shared_ptr<void> data(d_c_raw, deleter);

    bool c_requires_grad = a.requires_grad();

    return Tensor(a.shape(), a.strides(), a.dtype(), a.device(), data, 0, c_requires_grad, nullptr, std::nullopt);
}
