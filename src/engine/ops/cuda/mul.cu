#include "hip/hip_runtime.h"
#include "tensor.h"
#include "helpers.h"
#include "engine/ops/impl/mul.h"
#include "device.h"
#include <hip/hip_runtime.h>
#include <stdexcept>
#include "allocator/allocatorFactory.h"

#define CUDA_CHECK(err) { \
    hipError_t err_ = (err); \
    if (err_ != hipSuccess) { \
        throw std::runtime_error("CUDA Error: " + std::string(hipGetErrorString(err_))); \
    } \
}


__global__ void mul_kernel(float* c, float* a, float* b, size_t n) {
    size_t index = blockDim.x * blockIdx.x + threadIdx.x;
    size_t stride = gridDim.x * blockDim.x;

    for (int i = index; i < n; i += stride) {
        c[i] = a[i] * b[i];
    }
}

Tensor mul_gpu(const Tensor &a, const Tensor &b) {
    if (a.device().type != DeviceType::CUDA) { throw std::runtime_error("add_gpu can only operate on CUDA tensors."); }
    if (!a.is_contiguous()) {
        throw std::runtime_error("CUDA mul currently only supports contiguous tensors.");
    }

    bool c_requires_grad = a.requires_grad() || b.requires_grad();
    Tensor c(a.shape(), a.dtype(), deviceToString(a.device()), c_requires_grad);

    float* c_data = static_cast<float*>(c.data_ptr().get());
    float* a_data = static_cast<float*>(a.data_ptr().get());
    float* b_data = static_cast<float*>(b.data_ptr().get());
    size_t num_elements = a.numel();
    if (num_elements == 0) {
        return a;
    }

    int threadsPerBlock = 256;
    int blocksPerGrid = (num_elements + threadsPerBlock - 1) / threadsPerBlock;

    mul_kernel<<<blocksPerGrid, threadsPerBlock>>>(c_data, a_data, b_data, num_elements);


    CUDA_CHECK(hipGetLastError());

    std::vector<__int64_t> c_shape = a.shape();
    std::vector<__int64_t> c_strides = compute_strides_(c_shape);
    bool c_requries_grad = a.requires_grad() || b.requires_grad();

    auto allocator = AllocatorFactory::get(c.device());
    void* raw_ptr = allocator->allocate(num_elements);

    if (raw_ptr == nullptr) {
        throw std::runtime_error("Memory allocation failed for tensor on device cuda. The device might be out of memory.");
    }
    
    auto deleter = [allocator](void* ptr) { allocator->deallocate(ptr); };
    c.set_data_ptr(std::shared_ptr<void>(raw_ptr, deleter));

    return c;
}
