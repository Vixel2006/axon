#include "hip/hip_runtime.h"
#include "engine/ops.h"
#include "autograd/ops.h"
#include "tensor.h"
#include "helpers.h"
#include "utils.h"
#include "allocator/allocatorFactory.h"
#include <hip/hip_runtime.h>
#include <stdexcept>

__global__ void broadcast_mul_kernel(
    const float* a_data,
    const int64_t* a_strides,
    const float* b_data,
    const int64_t* b_strides,
    float* c_data,
    const int64_t* c_shape,
    int c_ndim,
    size_t num_elements)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_elements) {
        return;
    }

    int64_t temp_i = i;
    size_t a_offset = 0;
    size_t b_offset = 0;

    for (int d = c_ndim - 1; d >= 0; --d) {
        int64_t coord = temp_i % c_shape[d];
        temp_i /= c_shape[d];

        a_offset += coord * a_strides[d];
        b_offset += coord * b_strides[d];
    }

    c_data[i] = a_data[a_offset] * b_data[b_offset];
}

__global__ void scalar_mul_kernel(const float* __restrict__ a_data, float scalar, float* __restrict__ c_data, size_t num_elements) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (size_t i = index; i < num_elements; i += stride) {
      c_data[i] = a_data[i] * scalar;
    }
}

Tensor CudaOps::mul(const Tensor& a, float scalar) {
    if (a.device().type != DeviceType::CUDA) {
        throw std::runtime_error("CudaOps::mul can only operate on CUDA tensors.");
    }

    const size_t num_elements = a.numel();
    if (num_elements == 0) {
        return Tensor(a.shape(), a.dtype(), deviceToString(a.device()), false);
    }
    const size_t data_size = num_elements * sizeof(float);

    const float* d_a = static_cast<const float*>(a.raw_ptr());

    auto allocator = AllocatorFactory::get(a.device());
    void* d_c_raw = allocator->allocate(data_size);
    if (!d_c_raw) {
        throw std::runtime_error("Failed to allocate CUDA memory for output tensor via AllocatorFactory.");
    }
    float* d_c = static_cast<float*>(d_c_raw);

    const int threadsPerBlock = 256;
    const int blocksPerGrid = (num_elements + threadsPerBlock - 1) / threadsPerBlock;

    scalar_mul_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, scalar, d_c, num_elements);
    CUDA_CHECK(hipGetLastError());

    auto deleter = [allocator](void *ptr) { allocator->deallocate(ptr); };
    std::shared_ptr<void> data(d_c_raw, deleter);

    bool c_requires_grad = a.requires_grad();
    Tensor t = Tensor(a.shape(), a.strides(), a.dtype(), a.device(), data, 0, c_requires_grad, nullptr, std::nullopt);

    if (c_requires_grad) {
      t.set_ctx({a}, CudaAutograd::mul);
    }

    return t;
}

Tensor CudaOps::mul(const Tensor& a, const Tensor& b) {
    if (a.device().type != DeviceType::CUDA || b.device().type != DeviceType::CUDA) {
        throw std::runtime_error("Input tensors for CudaOps::mul must be on the CUDA device.");
    }

    std::vector<int64_t> c_shape = compute_broadcast_shape(a.shape(), b.shape());
    size_t num_elements = std::accumulate(c_shape.begin(), c_shape.end(), 1, std::multiplies<int64_t>());
    int c_ndim = c_shape.size();

    if (num_elements == 0) {
        return Tensor(c_shape, a.dtype(), deviceToString(a.device()), false);
    }

    Tensor a_broad = a.broadcast(c_shape);
    Tensor b_broad = b.broadcast(c_shape);

    const float* d_a = static_cast<const float*>(a_broad.raw_ptr());
    const float* d_b = static_cast<const float*>(b_broad.raw_ptr());

    auto allocator = AllocatorFactory::get(a.device());
    void* d_c_raw = allocator->allocate(num_elements * sizeof(float));
    float* d_c = static_cast<float*>(d_c_raw);

    int64_t *d_a_strides, *d_b_strides, *d_c_shape;
    CUDA_CHECK(hipMalloc(&d_a_strides, c_ndim * sizeof(int64_t)));
    CUDA_CHECK(hipMalloc(&d_b_strides, c_ndim * sizeof(int64_t)));
    CUDA_CHECK(hipMalloc(&d_c_shape, c_ndim * sizeof(int64_t)));

    CUDA_CHECK(hipMemcpy(d_a_strides, a_broad.strides().data(), c_ndim * sizeof(int64_t), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b_strides, b_broad.strides().data(), c_ndim * sizeof(int64_t), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_c_shape, c_shape.data(), c_ndim * sizeof(int64_t), hipMemcpyHostToDevice));

    const int threadsPerBlock = 256;
    const int blocksPerGrid = (num_elements + threadsPerBlock - 1) / threadsPerBlock;
    broadcast_mul_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_a, d_a_strides,
        d_b, d_b_strides,
        d_c, d_c_shape, c_ndim, num_elements
    );
    CUDA_CHECK(hipGetLastError());

    CUDA_CHECK(hipFree(d_a_strides));
    CUDA_CHECK(hipFree(d_b_strides));
    CUDA_CHECK(hipFree(d_c_shape));

    auto deleter = [allocator](void* ptr) { allocator->deallocate(ptr); };
    std::shared_ptr<void> data(d_c_raw, deleter);

    bool c_requires_grad = a.requires_grad() || b.requires_grad();
    Tensor t = Tensor(c_shape, a.dtype(), deviceToString(a.device()), c_requires_grad);
    t.set_data_ptr(data);

    if (c_requires_grad) {
      t.set_ctx({a, b}, CudaAutograd::mul);
    }

    return t;
}

