#include "tensor.h"
#include "engine/ops/impl/matmul.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <cstdint>
#include <helpers.h>
#include <cstdio>

#define CUDA_CHECK(call)                                                      \
do {                                                                          \
    hipError_t err = call;                                                   \
    if (err != hipSuccess) {                                                 \
        fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, \
                hipGetErrorString(err));                                     \
    }                                                                         \
} while (0)

#define TILE_DIM 32

Tensor matmul_gpu(const Tensor& a, const Tensor& b) {
    // TODO: Implement matmul_gpu
}
