#include "hip/hip_runtime.h"
#include "engine/ops.h"
#include "autograd/ops.h"
#include "tensor.h"
#include "helpers.h"
#include "utils.h"
#include "allocator/allocatorFactory.h"
#include <hip/hip_runtime.h>
#include <stdexcept>

__global__ void element_wise_add_kernel(const float* a_data, const float* b_data, float* c_data, size_t num_elements) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = index; i < num_elements; i += stride) {
        c_data[i] = a_data[i] + b_data[i];
    }
}

__global__ void scalar_add_kernel(const float* a_data, float scalar, float* c_data, size_t num_elements) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = index; i < num_elements; i += stride) {
        c_data[i] = a_data[i] + scalar;
    }
}

Tensor CudaOps::add(const Tensor &a, float scalar) {
    if (a.device().type != DeviceType::CUDA) {
        throw std::runtime_error("Input tensors for CudaOps::add must be on the CUDA device.");
    }

    const size_t num_elements = a.numel();
    if (num_elements == 0) {
        return Tensor(a.shape(), a.dtype(), deviceToString(a.device()), false);
    }
    const size_t data_size = num_elements * sizeof(float);

    const float* d_a = static_cast<const float*>(a.raw_ptr());

    auto allocator = AllocatorFactory::get(a.device());
    void* d_c_raw = allocator->allocate(data_size);
    if (!d_c_raw) {
        throw std::runtime_error("Failed to allocate CUDA memory for output tensor via AllocatorFactory.");
    }
    float* d_c = static_cast<float*>(d_c_raw);

    const int threadsPerBlock = 256;
    const int blocksPerGrid = (num_elements + threadsPerBlock - 1) / threadsPerBlock;

    scalar_add_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, scalar, d_c, num_elements);
    CUDA_CHECK(hipGetLastError());

    auto deleter = [allocator](void *ptr) { allocator->deallocate(ptr); };
    std::shared_ptr<void> data(d_c_raw, deleter);

    bool c_requires_grad = a.requires_grad();
    Tensor t = Tensor(a.shape(), a.strides(), a.dtype(), a.device(), data, 0, c_requires_grad, nullptr, std::nullopt);
   
    if (c_requires_grad) {
      t.set_ctx({a}, CudaAutograd::add);
    }

    return t;
}

Tensor CudaOps::add(const Tensor &a, const Tensor &b) {
    if (a.shape() != b.shape()) {
        throw std::runtime_error("Tensor shapes are mismatched for addition.");
    }
    if (a.device().type != DeviceType::CUDA || b.device().type != DeviceType::CUDA) {
        throw std::runtime_error("Input tensors for CudaOps::add must be on the CUDA device.");
    }

    const size_t num_elements = a.numel();
    if (num_elements == 0) {
        return Tensor(a.shape(), a.dtype(), deviceToString(a.device()), false);
    }
    const size_t data_size = num_elements * sizeof(float);

    const float* d_a = static_cast<const float*>(a.raw_ptr());
    const float* d_b = static_cast<const float*>(b.raw_ptr());

    auto allocator = AllocatorFactory::get(a.device());
    void* d_c_raw = allocator->allocate(data_size);
    if (!d_c_raw) {
        throw std::runtime_error("Failed to allocate CUDA memory for output tensor via AllocatorFactory.");
    }
    float* d_c = static_cast<float*>(d_c_raw);

    const int threadsPerBlock = 256;
    const int blocksPerGrid = (num_elements + threadsPerBlock - 1) / threadsPerBlock;

    element_wise_add_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, num_elements);
    CUDA_CHECK(hipGetLastError());

    auto deleter = [allocator](void *ptr) { allocator->deallocate(ptr); };
    std::shared_ptr<void> data(d_c_raw, deleter);

    bool c_requires_grad = a.requires_grad() || b.requires_grad();
    Tensor t = Tensor(a.shape(), a.strides(), a.dtype(), a.device(), data, 0, c_requires_grad, nullptr, std::nullopt);
   
    if (c_requires_grad) {
      t.set_ctx({a, b}, CudaAutograd::add);
    }

    return t;
}
