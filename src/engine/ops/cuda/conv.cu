#include "hip/hip_runtime.h"
#include "engine/ops.h"
#include "tensor.h"
#include "helpers.h"
#include "utils.h"
#include "autograd/ops.h"

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdexcept>
#include <vector>

__global__ void complex_mult_accumulate_kernel(hipfftComplex* accumulator, const hipfftComplex* a, const hipfftComplex* b, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float ar = a[idx].x;
        float ai = a[idx].y;
        float br = b[idx].x;
        float bi = b[idx].y;
        
        // C = A * B
        float cr = ar * br - ai * bi;
        float ci = ar * bi + ai * br;

        // Accumulator += C
        atomicAdd(&accumulator[idx].x, cr);
        atomicAdd(&accumulator[idx].y, ci);
    }
}


Tensor CudaOps::conv2d(const Tensor& a, const Tensor& kernel, int stride, int padding) {
    const std::vector<int64_t>& in_shape = a.shape();
    const std::vector<int64_t>& kernel_shape = kernel.shape();

    const int64_t N = in_shape[0];
    const int64_t C_in = in_shape[1];
    const int64_t H_in = in_shape[2];
    const int64_t W_in = in_shape[3];

    const int64_t C_out = kernel_shape[0];
    const int64_t H_k = kernel_shape[2];
    const int64_t W_k = kernel_shape[3];

    const int64_t H_out = (H_in + 2 * padding - H_k) / stride + 1;
    const int64_t W_out = (W_in + 2 * padding - W_k) / stride + 1;

    const int64_t H_fft = next_power_of_2(H_in + H_k - 1);
    const int64_t W_fft = next_power_of_2(W_in + W_k - 1);
    const int64_t W_fft_complex = (W_fft / 2) + 1;

    const size_t fft_real_size = W_fft * H_fft * sizeof(float);
    const size_t fft_complex_size = W_fft_complex * H_fft * sizeof(hipfftComplex);
    const int    complex_elements = W_fft_complex * H_fft;

    Tensor out({N, C_out, H_out, W_out}, a.dtype(), deviceToString(a.device()), a.requires_grad());
    float* d_out_full = static_cast<float*>(out.data_ptr().get());

    hipfftHandle plan_r2c, plan_c2r;
    checkCufftErrors(hipfftPlan2d(&plan_r2c, H_fft, W_fft, HIPFFT_R2C));
    checkCufftErrors(hipfftPlan2d(&plan_c2r, H_fft, W_fft, HIPFFT_C2R));

    float* d_kernel_all = nullptr;
    hipfftComplex* d_kernels_fft_all = nullptr;
    CUDA_CHECK(hipMalloc(&d_kernel_all, kernel.numel()));
    CUDA_CHECK(hipMalloc(&d_kernels_fft_all, C_out * C_in * fft_complex_size));
    CUDA_CHECK(hipMemcpy(d_kernel_all, kernel.data_ptr().get(), kernel.numel(), hipMemcpyHostToDevice));

    float* h_kernel_slice = new float[H_k * W_k];
    float* d_padded_temp;
    CUDA_CHECK(hipMalloc(&d_padded_temp, fft_real_size));
    
    for (int64_t c_out = 0; c_out < C_out; ++c_out) {
        for (int64_t c_in = 0; c_in < C_in; ++c_in) {
            const int64_t kernel_channel_offset = (c_out * C_in + c_in) * H_k * W_k;
            const int64_t kernel_fft_offset = (c_out * C_in + c_in) * complex_elements;

            pad_kernel<<<dim3((W_fft+15)/16, (H_fft+15)/16), dim3(16,16)>>>(
                d_kernel_all + kernel_channel_offset, d_padded_temp, W_k, H_k, W_fft, H_fft);
            
            checkCufftErrors(hipfftExecR2C(plan_r2c, (hipfftReal*)d_padded_temp, d_kernels_fft_all + kernel_fft_offset));
        }
    }
    delete[] h_kernel_slice;
    CUDA_CHECK(hipFree(d_padded_temp));


    float* d_input_all = nullptr;
    float* d_padded_input = nullptr;
    hipfftComplex* d_input_fft = nullptr;
    hipfftComplex* d_acc_fft = nullptr;
    float* d_conv_result_padded = nullptr;
    CUDA_CHECK(hipMalloc(&d_input_all, a.numel()));
    CUDA_CHECK(hipMalloc(&d_padded_input, fft_real_size));
    CUDA_CHECK(hipMalloc(&d_input_fft, fft_complex_size));
    CUDA_CHECK(hipMalloc(&d_acc_fft, fft_complex_size));
    CUDA_CHECK(hipMalloc(&d_conv_result_padded, fft_real_size));
    CUDA_CHECK(hipMemcpy(d_input_all, a.data_ptr().get(), a.numel(), hipMemcpyHostToDevice));


    for (int64_t n = 0; n < N; ++n) {
        for (int64_t c_out = 0; c_out < C_out; ++c_out) {
            CUDA_CHECK(hipMemset(d_acc_fft, 0, fft_complex_size));

            for (int64_t c_in = 0; c_in < C_in; ++c_in) {
                const float* d_current_input_slice = d_input_all + (n * C_in + c_in) * H_in * W_in;
                const hipfftComplex* d_current_kernel_fft = d_kernels_fft_all + (c_out * C_in + c_in) * complex_elements;

                pad_kernel<<<dim3((W_fft+15)/16, (H_fft+15)/16), dim3(16,16)>>>(
                    d_current_input_slice, d_padded_input, W_in, H_in, W_fft, H_fft);

                checkCufftErrors(hipfftExecR2C(plan_r2c, (hipfftReal*)d_padded_input, d_input_fft));

                int threads = 256;
                int blocks = (complex_elements + threads - 1) / threads;
                complex_mult_accumulate_kernel<<<blocks, threads>>>(
                    d_acc_fft, d_input_fft, d_current_kernel_fft, complex_elements);
            }

            checkCufftErrors(hipfftExecC2R(plan_c2r, d_acc_fft, (hipfftReal*)d_conv_result_padded));
            
            float* d_current_out_slice = d_out_full + (n * C_out + c_out) * H_out * W_out;
            dim3 crop_block_dim(16, 16);
            dim3 crop_grid_dim((W_out + 15) / 16, (H_out + 15) / 16);
            crop_and_stride_kernel<<<crop_grid_dim, crop_block_dim>>>(
                d_conv_result_padded, d_current_out_slice, W_fft, W_out, H_out, W_k, H_k, stride, padding);
        }
    }
    
    CUDA_CHECK(hipFree(d_kernel_all));
    CUDA_CHECK(hipFree(d_kernels_fft_all));
    CUDA_CHECK(hipFree(d_input_all));
    CUDA_CHECK(hipFree(d_padded_input));
    CUDA_CHECK(hipFree(d_input_fft));
    CUDA_CHECK(hipFree(d_acc_fft));
    CUDA_CHECK(hipFree(d_conv_result_padded));
    checkCufftErrors(hipfftDestroy(plan_r2c));
    checkCufftErrors(hipfftDestroy(plan_c2r));
    
    return out;
}

