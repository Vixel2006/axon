#include "engine/ops.h"
#include "tensor.h"
#include "helpers.h"
#include <hip/hip_runtime.h>
#include <stdexcept>

#define CUDA_CHECK(call)                                                    \
    do {                                                                    \
        hipError_t err = call;                                             \
        if (err != hipSuccess) {                                           \
            throw std::runtime_error(std::string("CUDA Error in " #call " : ") + \
                                     hipGetErrorString(err));              \
        }                                                                   \
    } while (0)


Tensor CudaOps::sum(const Tensor &a, int dim, bool keepdim) {
}
