#include "hip/hip_runtime.h"
#include "engine/ops.h"
#include "autograd/ops.h"
#include "tensor.h"
#include "helpers.h"
#include "utils.h"
#include "allocator/allocatorFactory.h"
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <vector>

__global__ void sum_reduction_kernel(
    const float* in_data,
    float* out_data,
    const int64_t* d_in_strides,
    const int64_t* d_out_strides,
    const int64_t* d_in_shape,
    int ndim,
    int reduction_dim,
    bool keepdim,
    size_t num_output_elements
) {
    extern __shared__ float sdata[];

    int64_t output_idx = blockIdx.x;
    if (output_idx >= num_output_elements) {
        return;
    }

    const int64_t reduction_size = d_in_shape[reduction_dim];
    const int64_t reduction_stride = d_in_strides[reduction_dim];

    int64_t start_in_offset = 0;
    int64_t temp_i = output_idx;
    int out_dim_idx = 0;
    for (int in_dim_idx = 0; in_dim_idx < ndim; ++in_dim_idx) {
        if (in_dim_idx == reduction_dim) {
            if (keepdim) {
                out_dim_idx++;
            }
            continue;
        }

        const int64_t coord = temp_i / d_out_strides[out_dim_idx];
        start_in_offset += coord * d_in_strides[in_dim_idx];
        temp_i %= d_out_strides[out_dim_idx];
        out_dim_idx++;
    }

    float sum = 0.0f;
    for (int64_t j = threadIdx.x; j < reduction_size; j += blockDim.x) {
        sum += in_data[start_in_offset + j * reduction_stride];
    }
    sdata[threadIdx.x] = sum;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            sdata[threadIdx.x] += sdata[threadIdx.x + s];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        out_data[output_idx] = sdata[0];
    }
}

Tensor CudaOps::sum(const Tensor &a, int dim, bool keepdim) {
    if (a.device().type != DeviceType::CUDA) {
        throw std::runtime_error("Input tensor for CudaOps::sum must be on the CUDA device.");
    }
    int ndim = a.ndim();
    if (dim < 0) {
        dim += ndim;
    }
    if (dim < 0 || dim >= ndim) {
        throw std::runtime_error("Reduction dimension is out of bounds.");
    }

    std::vector<int64_t> new_shape = reduce_shape(a.shape(), dim, keepdim);
    bool result_requires_grad = a.requires_grad();
    Tensor result(new_shape, a.dtype(), deviceToString(a.device()), result_requires_grad);

    if (a.numel() == 0) {
        return result;
    }
    const size_t num_output_elements = result.numel();

    const float* d_a = static_cast<const float*>(a.raw_ptr());
    float* d_result = static_cast<float*>(result.raw_ptr());

    const auto& in_shape_vec = a.shape();
    const auto& in_strides_vec = a.strides();
    const auto& out_strides_vec = result.strides();

    int64_t* d_in_shape;
    int64_t* d_in_strides;
    int64_t* d_out_strides;

    CUDA_CHECK(hipMalloc(&d_in_shape, in_shape_vec.size() * sizeof(int64_t)));
    CUDA_CHECK(hipMalloc(&d_in_strides, in_strides_vec.size() * sizeof(int64_t)));
    CUDA_CHECK(hipMalloc(&d_out_strides, out_strides_vec.size() * sizeof(int64_t)));

    CUDA_CHECK(hipMemcpy(d_in_shape, in_shape_vec.data(), in_shape_vec.size() * sizeof(int64_t), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_in_strides, in_strides_vec.data(), in_strides_vec.size() * sizeof(int64_t), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_out_strides, out_strides_vec.data(), out_strides_vec.size() * sizeof(int64_t), hipMemcpyHostToDevice));

    const int threadsPerBlock = 256;
    const int blocksPerGrid = num_output_elements;
    const size_t shmem_size = threadsPerBlock * sizeof(float);

    sum_reduction_kernel<<<blocksPerGrid, threadsPerBlock, shmem_size>>>(
        d_a,
        d_result,
        d_in_strides,
        d_out_strides,
        d_in_shape,
        ndim,
        dim,
        keepdim,
        num_output_elements
    );
    CUDA_CHECK(hipGetLastError());

    CUDA_CHECK(hipFree(d_in_shape));
    CUDA_CHECK(hipFree(d_in_strides));
    CUDA_CHECK(hipFree(d_out_strides));

    if (result_requires_grad) {
      result.set_ctx({a}, CudaAutograd::sum);
    }


    return result;
}
