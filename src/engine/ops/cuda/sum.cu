#include "hip/hip_runtime.h"
#include "engine/ops.h"
#include "tensor.h"
#include "helpers.h"
#include "utils.h"
#include <hip/hip_runtime.h>
#include <stdexcept>

<<<<<<< Updated upstream
=======
__global__ void full_reduction_sum_kernel(
    const float* in_data,
    float* out_data,
    size_t num_elements
) {
    extern __shared__ float sdata[];

    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    float local_sum = 0.0f;
    for (size_t i = index; i < num_elements; i += stride) {
        local_sum += in_data[i];
    }
    sdata[tid] = local_sum;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(out_data, sdata[0]);
    }
}

__global__ void sum_reduction_kernel(
    const float* in_data,
    float* out_data,
    const int64_t* d_in_strides,
    const int64_t* d_out_strides,
    const int64_t* d_in_shape,
    int ndim,
    int reduction_dim,
    bool keepdim,
    size_t num_output_elements
) {
    extern __shared__ float sdata[];

    int64_t output_idx = blockIdx.x;
    if (output_idx >= num_output_elements) {
        return;
    }

    const int64_t reduction_size = d_in_shape[reduction_dim];
    const int64_t reduction_stride = d_in_strides[reduction_dim];

    int64_t start_in_offset = 0;
    int64_t temp_i = output_idx;
    int out_dim_idx = 0;
    for (int in_dim_idx = 0; in_dim_idx < ndim; ++in_dim_idx) {
        if (in_dim_idx == reduction_dim) {
            if (keepdim) {
                out_dim_idx++;
            }
            continue;
        }

        const int64_t coord = temp_i / d_out_strides[out_dim_idx];
        start_in_offset += coord * d_in_strides[in_dim_idx];
        temp_i %= d_out_strides[out_dim_idx];
        out_dim_idx++;
    }

    float sum = 0.0f;
    for (int64_t j = threadIdx.x; j < reduction_size; j += blockDim.x) {
        sum += in_data[start_in_offset + j * reduction_stride];
    }
    sdata[threadIdx.x] = sum;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            sdata[threadIdx.x] += sdata[threadIdx.x + s];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        out_data[output_idx] = sdata[0];
    }
}
>>>>>>> Stashed changes

Tensor CudaOps::sum(const Tensor &a) {
    if (a.device().type != DeviceType::CUDA) {
        throw std::runtime_error("Input tensor for CudaOps::sum must be on the CUDA device.");
    }

    // The result of a full reduction is a scalar tensor.
    std::vector<int64_t> new_shape = {1};
    bool result_requires_grad = a.requires_grad();
    Tensor result(new_shape, a.dtype(), deviceToString(a.device()), result_requires_grad);

    const size_t num_elements = a.numel();
    if (num_elements == 0) {
        float* d_result = static_cast<float*>(result.raw_ptr());
        CUDA_CHECK(hipMemset(d_result, 0, sizeof(float)));
        return result;
    }

    const float* d_a = static_cast<const float*>(a.raw_ptr());
    float* d_result = static_cast<float*>(result.raw_ptr());

    CUDA_CHECK(hipMemset(d_result, 0, sizeof(float)));

    const int threadsPerBlock = 256;
    const int blocksPerGrid = std::min(
        (int)((num_elements + threadsPerBlock - 1) / threadsPerBlock),
        4096
    );
    const size_t shmem_size = threadsPerBlock * sizeof(float);

    full_reduction_sum_kernel<<<blocksPerGrid, threadsPerBlock, shmem_size>>>(
        d_a, d_result, num_elements
    );
    CUDA_CHECK(hipGetLastError());

    if (result_requires_grad) {
      result.set_ctx({a}, CudaAutograd::sum);
    }

    return result;
}

Tensor CudaOps::sum(const Tensor &a, int dim, bool keepdim) {
}
