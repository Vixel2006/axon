#include "engine/ops.h"
#include "tensor.h"
#include "helpers.h" // For AlignedDeleter
#include <hip/hip_runtime.h>
#include <stdexcept>

// Macro for robust CUDA error checking
#define CUDA_CHECK(call)                                                    \
    do {                                                                    \
        hipError_t err = call;                                             \
        if (err != hipSuccess) {                                           \
            throw std::runtime_error(std::string("CUDA Error in " #call " : ") + \
                                     hipGetErrorString(err));              \
        }                                                                   \
    } while (0)


Tensor CudaOps::sum(const Tensor &a, int dim, bool keepdim) {
}
