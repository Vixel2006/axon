#include "hip/hip_runtime.h"
#include "engine/ops.h"
#include "autograd/ops.h"
#include "tensor.h"
#include "helpers.h"
#include "utils.h"
#include "allocator/allocatorFactory.h"
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <vector>

__global__ void full_reduction_mean_kernel(
    const float* in_data,
    float* out_data,
    unsigned int* d_finished_blocks,
    size_t num_elements
) {
    extern __shared__ float sdata[];
    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    float local_sum = 0.0f;
    for (size_t i = index; i < num_elements; i += stride) {
        local_sum += in_data[i];
    }
    sdata[tid] = local_sum;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(out_data, sdata[0]);

        unsigned int finished_count = atomicInc(d_finished_blocks, gridDim.x);

        if (finished_count == gridDim.x - 1) {
            out_data[0] /= static_cast<float>(num_elements);
        }
    }
}

__global__ void mean_reduction_kernel(
    const float* in_data,
    float* out_data,
    const int64_t* d_in_strides,
    const int64_t* d_out_strides,
    const int64_t* d_in_shape,
    int ndim,
    int reduction_dim,
    bool keepdim,
    size_t num_output_elements
) {
    extern __shared__ float sdata[];

    int64_t output_idx = blockIdx.x;
    if (output_idx >= num_output_elements) {
        return;
    }

    const int64_t reduction_size = d_in_shape[reduction_dim];
    const int64_t reduction_stride = d_in_strides[reduction_dim];

    int64_t start_in_offset = 0;
    int64_t temp_i = output_idx;
    int out_dim_idx = 0;
    for (int in_dim_idx = 0; in_dim_idx < ndim; ++in_dim_idx) {
        if (in_dim_idx == reduction_dim) {
            if (keepdim) {
                out_dim_idx++;
            }
            continue;
        }

        const int64_t coord = temp_i / d_out_strides[out_dim_idx];
        start_in_offset += coord * d_in_strides[in_dim_idx];
        temp_i %= d_out_strides[out_dim_idx];
        out_dim_idx++;
    }

    float sum = 0.0f;
    for (int64_t j = threadIdx.x; j < reduction_size; j += blockDim.x) {
        sum += in_data[start_in_offset + j * reduction_stride];
    }
    sdata[threadIdx.x] = sum;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            sdata[threadIdx.x] += sdata[threadIdx.x + s];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        if (reduction_size > 0) {
            out_data[output_idx] = sdata[0] / reduction_size;
        } else {
            out_data[output_idx] = 0.0f;
        }
    }
}

Tensor CudaOps::mean(const Tensor &a) {
    if (a.device().type != DeviceType::CUDA) {
        throw std::runtime_error("Input tensor must be on CUDA device.");
    }
    std::vector<int64_t> new_shape = {1};
    Tensor result(new_shape, a.dtype(), deviceToString(a.device()), a.requires_grad());

    const size_t num_elements = a.numel();
    if (num_elements == 0) { return result; }

    const float* d_a = static_cast<const float*>(a.raw_ptr());
    float* d_result = static_cast<float*>(result.raw_ptr());

    unsigned int* d_finished_blocks;
    CUDA_CHECK(hipMalloc(&d_finished_blocks, sizeof(unsigned int)));
    CUDA_CHECK(hipMemset(d_finished_blocks, 0, sizeof(unsigned int)));
    CUDA_CHECK(hipMemset(d_result, 0, sizeof(float)));

    const int threadsPerBlock = 256;
    const int blocksPerGrid = std::min((int)((num_elements + threadsPerBlock - 1) / threadsPerBlock), 4096);
    const size_t shmem_size = threadsPerBlock * sizeof(float);

    full_reduction_mean_kernel<<<blocksPerGrid, threadsPerBlock, shmem_size>>>(
        d_a,
        d_result,
        d_finished_blocks,
        num_elements
    );
    CUDA_CHECK(hipGetLastError());

    CUDA_CHECK(hipFree(d_finished_blocks));

    if (a.requires_grad()) {
        result.set_ctx({a}, CudaAutograd::mean);
    }

    return result;
}

Tensor CudaOps::mean(const Tensor &a, int dim, bool keepdim) {
    if (a.device().type != DeviceType::CUDA) {
        throw std::runtime_error("Input tensor for CudaOps::mean must be on the CUDA device.");
    }
    int ndim = a.ndim();
    if (dim < 0) {
        dim += ndim;
    }
    if (dim < 0 || dim >= ndim) {
        throw std::runtime_error("Reduction dimension is out of bounds.");
    }

    std::vector<int64_t> new_shape = reduce_shape(a.shape(), dim, keepdim);
    bool result_requires_grad = a.requires_grad();
    Tensor result(new_shape, a.dtype(), deviceToString(a.device()), result_requires_grad);

    if (a.numel() == 0) {
        return result;
    }
    const size_t num_output_elements = result.numel();

    const float* d_a = static_cast<const float*>(a.raw_ptr());
    float* d_result = static_cast<float*>(result.raw_ptr());

    const auto& in_shape_vec = a.shape();
    const auto& in_strides_vec = a.strides();
    const auto& out_strides_vec = result.strides();

    int64_t* d_in_shape;
    int64_t* d_in_strides;
    int64_t* d_out_strides;

    CUDA_CHECK(hipMalloc(&d_in_shape, in_shape_vec.size() * sizeof(int64_t)));
    CUDA_CHECK(hipMalloc(&d_in_strides, in_strides_vec.size() * sizeof(int64_t)));
    CUDA_CHECK(hipMalloc(&d_out_strides, out_strides_vec.size() * sizeof(int64_t)));

    CUDA_CHECK(hipMemcpy(d_in_shape, in_shape_vec.data(), in_shape_vec.size() * sizeof(int64_t), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_in_strides, in_strides_vec.data(), in_strides_vec.size() * sizeof(int64_t), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_out_strides, out_strides_vec.data(), out_strides_vec.size() * sizeof(int64_t), hipMemcpyHostToDevice));

    const int threadsPerBlock = 256;
    const int blocksPerGrid = num_output_elements;
    const size_t shmem_size = threadsPerBlock * sizeof(float);

    mean_reduction_kernel<<<blocksPerGrid, threadsPerBlock, shmem_size>>>(
        d_a,
        d_result,
        d_in_strides,
        d_out_strides,
        d_in_shape,
        ndim,
        dim,
        keepdim,
        num_output_elements
    );
    CUDA_CHECK(hipGetLastError());

    CUDA_CHECK(hipFree(d_in_shape));
    CUDA_CHECK(hipFree(d_in_strides));
    CUDA_CHECK(hipFree(d_out_strides));

    if (result_requires_grad) {
      result.set_ctx({a}, CudaAutograd::mean);
    }

    return result;
}

