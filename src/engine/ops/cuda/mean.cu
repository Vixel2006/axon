#include "hip/hip_runtime.h"
#include "engine/ops.h"
#include "tensor.h"
#include "helpers.h"
#include <hip/hip_runtime.h>
#include <stdexcept>

<<<<<<< Updated upstream
#define CUDA_CHECK(call)                                                    \
    do {                                                                    \
        hipError_t err = call;                                             \
        if (err != hipSuccess) {                                           \
            throw std::runtime_error(std::string("CUDA Error in " #call " : ") + \
                                     hipGetErrorString(err));              \
        }                                                                   \
    } while (0)

struct AlignedDeleter {
    void operator()(void* ptr) const {
        #ifdef _MSC_VER
        _aligned_free(ptr);
        #else
        free(ptr);
        #endif
    }
};

__global__ void sum_reduce_kernel_for_mean(const float* input_data, float* output_data, size_t num_elements) {
=======
__global__ void full_reduction_mean_kernel(
    const float* in_data,
    float* out_data,
    unsigned int* d_finished_blocks,
    size_t num_elements
) {
    extern __shared__ float sdata[];

    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    float local_sum = 0.0f;
    for (size_t i = index; i < num_elements; i += stride) {
        local_sum += in_data[i];
    }
    sdata[tid] = local_sum;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(out_data, sdata[0]);

        unsigned int finished_count = atomicInc(d_finished_blocks, gridDim.x);

        if (finished_count == gridDim.x - 1) {
            out_data[0] /= static_cast<float>(num_elements);
        }
    }
}


__global__ void mean_reduction_kernel(
    const float* in_data,
    float* out_data,
    const int64_t* d_in_strides,
    const int64_t* d_out_strides,
    const int64_t* d_in_shape,
    int ndim,
    int reduction_dim,
    bool keepdim,
    size_t num_output_elements
) {
>>>>>>> Stashed changes
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = gridDim.x * blockDim.x;

    float my_sum = 0.0f;
    for (size_t j = i; j < num_elements; j += stride) {
        my_sum += input_data[j];
    }
    sdata[tid] = my_sum;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        output_data[blockIdx.x] = sdata[0];
    }
}

<<<<<<< Updated upstream
Tensor CudaOps::mean(const Tensor &a, int axis, bool keep_dims) {
=======
Tensor CudaOps::mean(const Tensor &a) {
    if (a.device().type != DeviceType::CUDA) {
        throw std::runtime_error("Input tensor must be on CUDA device.");
    }
    std::vector<int64_t> new_shape = {1};
    Tensor result(new_shape, a.dtype(), deviceToString(a.device()), a.requires_grad());

    const size_t num_elements = a.numel();
    if (num_elements == 0) { return result; }

    const float* d_a = static_cast<const float*>(a.raw_ptr());
    float* d_result = static_cast<float*>(result.raw_ptr());

    unsigned int* d_finished_blocks;
    CUDA_CHECK(hipMalloc(&d_finished_blocks, sizeof(unsigned int)));
    CUDA_CHECK(hipMemset(d_finished_blocks, 0, sizeof(unsigned int)));
    CUDA_CHECK(hipMemset(d_result, 0, sizeof(float)));

    const int threadsPerBlock = 256;
    const int blocksPerGrid = std::min((int)((num_elements + threadsPerBlock - 1) / threadsPerBlock), 4096);
    const size_t shmem_size = threadsPerBlock * sizeof(float);

    full_reduction_mean_kernel<<<blocksPerGrid, threadsPerBlock, shmem_size>>>(
        d_a,
        d_result,
        d_finished_blocks,
        num_elements
    );
    CUDA_CHECK(hipGetLastError());

    CUDA_CHECK(hipFree(d_finished_blocks));

    if (a.requires_grad()) {
        result.set_ctx({a}, CudaAutograd::mean);
    }

    return result;
}

Tensor CudaOps::mean(const Tensor &a, int dim, bool keepdim) {
    if (a.device().type != DeviceType::CUDA) {
        throw std::runtime_error("Input tensor for CudaOps::mean must be on the CUDA device.");
    }
    int ndim = a.ndim();
    if (dim < 0) {
        dim += ndim;
    }
    if (dim < 0 || dim >= ndim) {
        throw std::runtime_error("Reduction dimension is out of bounds.");
    }

    std::vector<int64_t> new_shape = reduce_shape(a.shape(), dim, keepdim);
    bool result_requires_grad = a.requires_grad();
    Tensor result(new_shape, a.dtype(), deviceToString(a.device()), result_requires_grad);

    if (a.numel() == 0) {
        return result;
    }
    const size_t num_output_elements = result.numel();

    const float* d_a = static_cast<const float*>(a.raw_ptr());
    float* d_result = static_cast<float*>(result.raw_ptr());

    const auto& in_shape_vec = a.shape();
    const auto& in_strides_vec = a.strides();
    const auto& out_strides_vec = result.strides();

    int64_t* d_in_shape;
    int64_t* d_in_strides;
    int64_t* d_out_strides;

    CUDA_CHECK(hipMalloc(&d_in_shape, in_shape_vec.size() * sizeof(int64_t)));
    CUDA_CHECK(hipMalloc(&d_in_strides, in_strides_vec.size() * sizeof(int64_t)));
    CUDA_CHECK(hipMalloc(&d_out_strides, out_strides_vec.size() * sizeof(int64_t)));

    CUDA_CHECK(hipMemcpy(d_in_shape, in_shape_vec.data(), in_shape_vec.size() * sizeof(int64_t), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_in_strides, in_strides_vec.data(), in_strides_vec.size() * sizeof(int64_t), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_out_strides, out_strides_vec.data(), out_strides_vec.size() * sizeof(int64_t), hipMemcpyHostToDevice));

    const int threadsPerBlock = 256;
    const int blocksPerGrid = num_output_elements;
    const size_t shmem_size = threadsPerBlock * sizeof(float);

    mean_reduction_kernel<<<blocksPerGrid, threadsPerBlock, shmem_size>>>(
        d_a,
        d_result,
        d_in_strides,
        d_out_strides,
        d_in_shape,
        ndim,
        dim,
        keepdim,
        num_output_elements
    );
    CUDA_CHECK(hipGetLastError());

    CUDA_CHECK(hipFree(d_in_shape));
    CUDA_CHECK(hipFree(d_in_strides));
    CUDA_CHECK(hipFree(d_out_strides));

    if (result_requires_grad) {
      result.set_ctx({a}, CudaAutograd::mean);
    }

    return result;
>>>>>>> Stashed changes
}

