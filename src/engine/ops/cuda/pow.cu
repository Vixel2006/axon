#include "hip/hip_runtime.h"
#include "engine/ops.h"
#include "tensor.h"
#include "helpers.h" // For AlignedDeleter
#include <hip/hip_runtime.h>
#include <stdexcept>

// Macro for robust CUDA error checking
#define CUDA_CHECK(call)                                                    \
    do {                                                                    \
        hipError_t err = call;                                             \
        if (err != hipSuccess) {                                           \
            throw std::runtime_error(std::string("CUDA Error in " #call " : ") + \
                                     hipGetErrorString(err));              \
        }                                                                   \
    } while (0)

// A custom deleter for host memory allocated with posix_memalign or _aligned_malloc
// It's good practice to define this in a central header (like helpers.h) to avoid re-definition.
struct AlignedDeleter {
    void operator()(void* ptr) const {
        #ifdef _MSC_VER
        _aligned_free(ptr);
        #else
        free(ptr);
        #endif
    }
};

/**
 * @brief CUDA kernel for element-wise power of a tensor with a SCALAR exponent.
 *
 * @param base_data Pointer to the device memory of the base tensor.
 * @param exponent The scalar float value for the exponent.
 * @param c_data Pointer to the device memory of the output tensor.
 * @param num_elements The total number of elements in the tensor.
 */
__global__ void pow_tensor_scalar_kernel(const float* base_data, const float exponent, float* c_data, size_t num_elements) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = index; i < num_elements; i += stride) {
        c_data[i] = powf(base_data[i], exponent); // Use powf for single-precision floats
    }
}

/**
 * @brief CUDA kernel for element-wise power of a tensor with a TENSOR exponent.
 *
 * @param base_data Pointer to the device memory of the base tensor.
 * @param exp_data Pointer to the device memory of the exponent tensor.
 * @param c_data Pointer to the device memory of the output tensor.
 * @param num_elements The total number of elements in the tensors.
 */
__global__ void pow_tensor_tensor_kernel(const float* base_data, const float* exp_data, float* c_data, size_t num_elements) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = index; i < num_elements; i += stride) {
        c_data[i] = powf(base_data[i], exp_data[i]);
    }
}


// Overload 1: For Tensor ^ Scalar
Tensor CudaOps::pow(const Tensor &base, float exponent) {
    if (base.device().type != DeviceType::CUDA) {
        throw std::runtime_error("Input tensor for CudaOps::pow must be on the CUDA device.");
    }

    const size_t num_elements = base.numel();
    if (num_elements == 0) {
        return Tensor({}, {}, base.dtype(), base.device(), nullptr, 0, false, nullptr, std::nullopt);
    }
    const size_t data_size = num_elements * sizeof(float);

    float *d_base, *d_c;
    CUDA_CHECK(hipMalloc(&d_base, data_size));
    CUDA_CHECK(hipMalloc(&d_c, data_size));

    CUDA_CHECK(hipMemcpy(d_base, base.data_ptr().get(), data_size, hipMemcpyHostToDevice));

    const int threadsPerBlock = 256;
    const int blocksPerGrid = (num_elements + threadsPerBlock - 1) / threadsPerBlock;

    pow_tensor_scalar_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_base, exponent, d_c, num_elements);
    CUDA_CHECK(hipGetLastError());

    void* c_data_raw = nullptr;
    #ifdef _MSC_VER
    c_data_raw = _aligned_malloc(data_size, 32);
    #else
    if (posix_memalign(&c_data_raw, 32, data_size) != 0) c_data_raw = nullptr;
    #endif

    if (!c_data_raw) {
        hipFree(d_base);
        hipFree(d_c);
        throw std::runtime_error("Failed to allocate aligned host memory for the output tensor.");
    }

    CUDA_CHECK(hipMemcpy(c_data_raw, d_c, data_size, hipMemcpyDeviceToHost));
    hipFree(d_base);
    hipFree(d_c);

    bool c_requires_grad = base.requires_grad();
    std::shared_ptr<void> data(c_data_raw, AlignedDeleter{});
    return Tensor(base.shape(), base.strides(), base.dtype(), base.device(), data, 0, c_requires_grad, nullptr, std::nullopt);
}


// Overload 2: For Tensor ^ Tensor
Tensor CudaOps::pow(const Tensor &base, const Tensor &exponent) {
    if (base.shape() != exponent.shape()) {
        throw std::runtime_error("Tensor shapes are mismatched for pow operation.");
    }
    if (base.device().type != DeviceType::CUDA || exponent.device().type != DeviceType::CUDA) {
        throw std::runtime_error("Input tensors for CudaOps::pow must be on the CUDA device.");
    }

    const size_t num_elements = base.numel();
    if (num_elements == 0) {
        return Tensor({}, {}, base.dtype(), base.device(), nullptr, 0, false, nullptr, std::nullopt);
    }
    const size_t data_size = num_elements * sizeof(float);

    float *d_base, *d_exp, *d_c;
    CUDA_CHECK(hipMalloc(&d_base, data_size));
    CUDA_CHECK(hipMalloc(&d_exp, data_size));
    CUDA_CHECK(hipMalloc(&d_c, data_size));

    CUDA_CHECK(hipMemcpy(d_base, base.data_ptr().get(), data_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_exp, exponent.data_ptr().get(), data_size, hipMemcpyHostToDevice));

    const int threadsPerBlock = 256;
    const int blocksPerGrid = (num_elements + threadsPerBlock - 1) / threadsPerBlock;

    pow_tensor_tensor_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_base, d_exp, d_c, num_elements);
    CUDA_CHECK(hipGetLastError());

    void* c_data_raw = nullptr;
    #ifdef _MSC_VER
    c_data_raw = _aligned_malloc(data_size, 32);
    #else
    if (posix_memalign(&c_data_raw, 32, data_size) != 0) c_data_raw = nullptr;
    #endif

    if (!c_data_raw) {
        hipFree(d_base);
        hipFree(d_exp);
        hipFree(d_c);
        throw std::runtime_error("Failed to allocate aligned host memory for the output tensor.");
    }

    CUDA_CHECK(hipMemcpy(c_data_raw, d_c, data_size, hipMemcpyDeviceToHost));
    hipFree(d_base);
    hipFree(d_exp);
    hipFree(d_c);

    bool c_requires_grad = base.requires_grad() || exponent.requires_grad();
    std::shared_ptr<void> data(c_data_raw, AlignedDeleter{});
    return Tensor(base.shape(), base.strides(), base.dtype(), base.device(), data, 0, c_requires_grad, nullptr, std::nullopt);
}
