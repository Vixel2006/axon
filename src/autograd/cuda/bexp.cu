#include "hip/hip_runtime.h"
#include "autograd/ops.h"
#include "tensor.h"
#include "utils.h"
#include <hip/hip_runtime.h>
#include <stdexcept>

__global__ void bexp_kernel(const float* out_grad_p,
                            const float* out_p,
                            float* a_grad_p,
                            size_t num_elements) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (size_t i = index; i < num_elements; i += stride) {
        const float grad = out_grad_p[i];
        const float out_val = out_p[i];
        
        a_grad_p[i] += grad * out_val;
    }
}

void CudaAutograd::exp(Tensor& out, std::vector<Tensor>& prev) {
    if (prev.size() != 1) {
        throw std::runtime_error("Exp backward operation requires exactly one previous tensor.");
    }

    Tensor t = out;
    Tensor& a = prev[0];
    
    if (!a.requires_grad()) {
        return;
    }

    const size_t num_elements = a.numel();
    if (num_elements == 0) {
        return;
    }

    const float* out_grad_p = static_cast<const float*>(t.grad_ptr().get());
    const float* out_p = static_cast<const float*>(t.data_ptr().get());
    float* a_grad_p = static_cast<float*>(a.grad_ptr().get());

    if (!out_grad_p || !out_p || !a_grad_p) {
        throw std::runtime_error("A data or gradient pointer is null in 'exp' backward pass (CUDA).");
    }

    const int threadsPerBlock = 256;
    const int blocksPerGrid = (num_elements + threadsPerBlock - 1) / threadsPerBlock;

    bexp_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        out_grad_p,
        out_p,
        a_grad_p,
        num_elements
    );

    CUDA_CHECK(hipGetLastError());
}
