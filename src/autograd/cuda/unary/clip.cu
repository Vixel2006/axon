#include "hip/hip_runtime.h"
#include "autograd/autograd_utils.h"
#include "autograd/cuda/unary/common.cuh"
#include "utils/indexing.cuh"

__global__ void clip_grad_kernel(const float* out_grad, const float* prev_data, float* prev_grad,
                                 float min_val, float max_val, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = idx; i < n; i += stride)
    {
        float x = prev_data[i];
        float mask = (prev_data[i] >= min_val) & (x <= max_val);
        prev_grad[i] += out_grad[i] * mask;
    }
}

__global__ void noncontig_clip_grad_kernel(const float* out_grad, const float* prev_data,
                                           float* prev_grad, float min_val, float max_val, int n,
                                           const int* shape, const int* strides, int ndim)
{
    int idx = blockIdx.x * blockIdx.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = idx; i < n; i += stride)
    {
        int in_idx = get_idx(shape, strides, ndim, i);
        float x = prev_data[in_idx];
        float mask = (prev_data[in_idx] >= min_val) & (x <= max_val);
        prev_grad[in_idx] += out_grad[i] * mask;
    }
}

void clip_grad_op_cuda(Tensor* out, Tensor** prev, int n_prev, void* extras)
{
    LOG_INFO("clip_grad_op_cuda: Entering function with n_prev=%d", n_prev);

    assert(out && "Output tensor cannot be NULL");
    assert(out->grad && "Output tensor gradient cannot be NULL");
    assert(out->grad->data && "Output tensor gradient data cannot be NULL");
    assert(out->grad->data->data && "Output tensor gradient data pointer cannot be NULL");
    assert(prev && "Previous tensors array cannot be NULL");
    assert(n_prev == 1 && "n_prev must be 1 for clip_grad_op_cuda");
    assert(extras && "Extras (ClipExtras) cannot be NULL");

    Tensor* a = prev[0];
    assert(a && "Input tensor 'a' cannot be NULL");
    assert(a->data && "Input tensor 'a' data cannot be NULL");
    assert(a->data->data && "Input tensor 'a' data pointer cannot be NULL");

    int N = numel(out->shape, out->ndim);

    int num_threads_per_block = 256;
    int num_blocks = (N + num_threads_per_block - 1) / num_threads_per_block;

    ClipExtras* clip_extras = (ClipExtras*) extras;
    float min_val = clip_extras->min_val;
    float max_val = clip_extras->max_val;

    if (prev[0]->requires_grad)
    {
        assert(a->grad && "Input tensor 'a' gradient cannot be NULL if requires_grad");
        assert(a->grad->data && "Input tensor 'a' gradient data cannot be NULL if requires_grad");
        assert(a->grad->data->data &&
               "Input tensor 'a' gradient data pointer cannot be NULL if requires_grad");
        if (is_contiguous(prev[0]))
        {
            clip_grad_kernel<<<num_blocks, num_threads_per_block>>>(
                out->grad->data->data, prev[0]->data->data, prev[0]->grad->data->data, min_val,
                max_val, N);
        }
        else
        {
            noncontig_clip_grad_kernel<<<num_blocks, num_threads_per_block>>>(
                out->grad->data->data, prev[0]->data->data, prev[0]->grad->data->data, min_val,
                max_val, N, prev[0]->shape, prev[0]->strides, prev[0]->ndim);
        }
        CHECK_CUDA();
    }
}
