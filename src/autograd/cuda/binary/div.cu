#include "hip/hip_runtime.h"
#include "autograd/cuda/binary/common.cuh"
#include "utils/indexing.cuh"

__global__ void numerator_div_grad_kernel(const float* out_grad, float* prev_grad,
                                          const float* denominator, int n)
{
    int idx = blockDim.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = idx; i < n; i += stride)
    {
        prev_grad[i] += out_grad[i] / (denominator[i] + 1e-7f);
    }
}

__global__ void noncontig_numerator_div_grad_kernel(const float* out_grad, float* prev_grad,
                                                    const float* denominator, int n,
                                                    const int* shape, const int* strides, int ndim)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = idx; i < n; i += stride)
    {
        int in_idx = get_idx(shape, strides, ndim, i);
        prev_grad[in_idx] += out_grad[i] / (denominator[in_idx] + 1e-7f);
    }
}

__global__ void denominator_div_grad_kernel(const float* out_grad, const float* out_data,
                                            float* prev_grad, float* denominator, int n)
{
    int idx = blockDim.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = idx; i < n; i += stride)
    {
        prev_grad[i] -= out_data[i] * out_grad[i] / (denominator[i] + 1e-7f);
    }
}

__global__ void noncontig_denominator_div_grad_kernel(const float* out_grad, const float* out_data,
                                                      float* prev_grad, float* denominator, int n,
                                                      const int* shape, const int* strides,
                                                      int ndim)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = idx; i < n; i += stride)
    {
        int in_idx = get_idx(shape, strides, ndim, i);
        prev_grad[in_idx] -= out_data[i] * out_grad[i] / (denominator[in_idx] + 1e-7f);
    }
}

__global__ void scalar_div_grad_kernel(const float* out_grad, float* prev_grad,
                                       float scalar_denominator, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = idx; i < n; i += stride)
    {
        prev_grad[i] += out_grad[i] / (scalar_denominator + 1e-7f);
    }
}

void div_grad_op_cuda(Tensor* out, Tensor** prev, int n_prev, void* extras)
{
    LOG_INFO("div_grad_op_cuda: Entering function with n_prev=%d", n_prev);

    assert(out && "Output tensor cannot be NULL");
    assert(out->grad && "Output tensor gradient cannot be NULL");
    assert(out->grad->data && "Output tensor gradient data cannot be NULL");
    assert(out->grad->data->data && "Output tensor gradient data pointer cannot be NULL");
    assert(prev && "Previous tensors array cannot be NULL");
    assert((n_prev == 1 || n_prev == 2) && "n_prev must be 1 or 2 for div_grad_op_cuda");

    int N = numel(out->shape, out->ndim);
    int num_threads_per_block = 256;
    int num_blocks = (N + num_threads_per_block - 1) / num_threads_per_block;

    if (n_prev == 1) // prev[0] / scalar
    {
        assert(extras && "Extras (scalar value) cannot be NULL for scalar division");
        float scalar_denominator = *((float*) extras);
        assert(prev[0] && "Previous tensor 0 cannot be NULL");
        assert(prev[0]->data && "Previous tensor 0 data cannot be NULL");
        assert(prev[0]->data->data && "Previous tensor 0 data pointer cannot be NULL");
        if (prev[0]->requires_grad)
        {
            assert(prev[0]->grad && "Previous tensor 0 gradient cannot be NULL if requires_grad");
            assert(prev[0]->grad->data &&
                   "Previous tensor 0 gradient data cannot be NULL if requires_grad");
            assert(prev[0]->grad->data->data &&
                   "Previous tensor 0 gradient data pointer cannot be NULL if requires_grad");
            scalar_div_grad_kernel<<<num_blocks, num_threads_per_block>>>(
                out->grad->data->data, prev[0]->grad->data->data, scalar_denominator, N);
            CHECK_CUDA();
        }
    }
    else
    {
        assert(prev[0] && "Previous tensor 0 (numerator) cannot be NULL");
        assert(prev[0]->data && "Previous tensor 0 (numerator) data cannot be NULL");
        assert(prev[0]->data->data && "Previous tensor 0 (numerator) data pointer cannot be NULL");
        assert(prev[1] && "Previous tensor 1 (denominator) cannot be NULL");
        assert(prev[1]->data && "Previous tensor 1 (denominator) data cannot be NULL");
        assert(prev[1]->data->data &&
               "Previous tensor 1 (denominator) data pointer cannot be NULL");

        if (prev[0]->requires_grad)
        {
            assert(prev[0]->grad && "Previous tensor 0 gradient cannot be NULL if requires_grad");
            assert(prev[0]->grad->data &&
                   "Previous tensor 0 gradient data cannot be NULL if requires_grad");
            assert(prev[0]->grad->data->data &&
                   "Previous tensor 0 gradient data pointer cannot be NULL if requires_grad");
            if (is_contiguous(prev[0]))
            {
                numerator_div_grad_kernel<<<num_blocks, num_threads_per_block>>>(
                    out->grad->data->data, prev[0]->grad->data->data, prev[1]->data->data, N);
            }
            else
            {
                noncontig_numerator_div_grad_kernel<<<num_blocks, num_threads_per_block>>>(
                    out->grad->data->data, prev[0]->grad->data->data, prev[1]->data->data, N,
                    prev[0]->shape, prev[0]->strides, prev[0]->ndim);
            }
            CHECK_CUDA();
        }

        if (prev[1]->requires_grad)
        {
            assert(prev[1]->grad && "Previous tensor 1 gradient cannot be NULL if requires_grad");
            assert(prev[1]->grad->data &&
                   "Previous tensor 1 gradient data cannot be NULL if requires_grad");
            assert(prev[1]->grad->data->data &&
                   "Previous tensor 1 gradient data pointer cannot be NULL if requires_grad");
            if (is_contiguous(prev[1]))
            {
                denominator_div_grad_kernel<<<num_blocks, num_threads_per_block>>>(
                    out->grad->data->data, out->data->data, prev[1]->grad->data->data,
                    prev[1]->data->data, N);
            }
            else
            {
                noncontig_denominator_div_grad_kernel<<<num_blocks, num_threads_per_block>>>(
                    out->grad->data->data, out->data->data, prev[1]->grad->data->data,
                    prev[1]->data->data, N, prev[1]->shape, prev[1]->strides, prev[1]->ndim);
            }
            CHECK_CUDA();
        }
    }
}
