#include "autograd/ops.h"
#include "tensor.h"
#include "helpers.h"
#include "utils.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdexcept>
#include <vector>

#include <hipblas.h>


void CudaAutograd::conv2d(Tensor& out, std::vector<Tensor>& prev, int stride, int padding) {
}
